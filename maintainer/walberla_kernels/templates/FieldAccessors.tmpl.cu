#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023-2024 The ESPResSo project
 * Copyright (C) 2020 The waLBerla project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

/**
 * @file
 * Lattice field accessors.
 * Adapted from the waLBerla source file
 * https://i10git.cs.fau.de/walberla/walberla/-/blob/a16141524c58ab88386e2a0f8fdd7c63c5edd704/python/lbmpy_walberla/templates/LatticeModel.tmpl.h
 */

#include <core/DataTypes.h>
#include <core/cell/Cell.h>
#include <core/cell/CellInterval.h>
#include <core/math/Matrix{{D}}.h>
#include <core/math/Vector{{D}}.h>

#include <field/iterators/IteratorMacros.h>

#include <cuda/FieldAccessor.h>
#include <cuda/FieldIndexing.h>
#include <cuda/GPUField.h>
#include <cuda/Kernel.h>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

#include <array>
#include <tuple>
#include <vector>

#ifdef __GNUC__
#define RESTRICT __restrict__
#elif _MSC_VER
#define RESTRICT __restrict
#else
#define RESTRICT
#endif

__device__ inline uint get_num_threads( uint3 gridDim, uint3 blockDim ) {
  return gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
}

__device__ inline uint getLinearIndexXYZF( uint3 blockIdx, uint3 threadIdx, uint3 gridDim, uint3 blockDim ) {
  auto const x = threadIdx.x;
  auto const y = blockIdx.x;
  auto const z = blockIdx.y;
  auto const f = blockIdx.z;
  auto const xSize = blockDim.x;
  auto const ySize = gridDim.x;
  auto const zSize = gridDim.y;
  return x                         +
         y * xSize                 +
         z * xSize * ySize         +
         f * xSize * ySize * zSize ;
}

__device__ inline uint getLinearIndexFZYX( uint3 blockIdx, uint3 threadIdx, uint3 gridDim, uint3 blockDim, uint fOffset ) {
  auto const x = threadIdx.x;
  auto const y = blockIdx.x;
  auto const z = blockIdx.y;
  auto const f = blockIdx.z;
  auto const ySize = gridDim.x;
  auto const zSize = gridDim.y;
  auto const fSize = fOffset;
  return f                         +
         z * fSize                 +
         y * fSize * zSize         +
         x * fSize * zSize * ySize ;
}

namespace walberla {
namespace {{namespace}} {
namespace accessor {

namespace Population
{
    __global__ void kernel_get_interval(
        cuda::FieldAccessor< {{dtype}} > pdf,
        {{dtype}} * RESTRICT const pop )
    {
        pdf.set( blockIdx, threadIdx );
        if (pdf.isValidPosition()) {
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, {{Q}}u);
            {% for i in range(Q) -%}
                pop[offset + {{i}}u] = pdf.get({{i}});
            {% endfor -%}
        }
    }

    __global__ void kernel_get(
        cuda::FieldAccessor< {{dtype}} > pdf,
        {{dtype}} * RESTRICT const pop )
    {
        pdf.set( blockIdx, threadIdx );
        if (pdf.isValidPosition()) {
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, {{Q}}u);
            {% for i in range(Q) -%}
                pop[{{i}}u] = pdf.get({{i}});
            {% endfor -%}
        }
    }

    __global__ void kernel_set_interval(
        cuda::FieldAccessor< {{dtype}} > pdf,
        const {{dtype}} * RESTRICT const pop )
    {
        pdf.set( blockIdx, threadIdx );
        if (pdf.isValidPosition()) {
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, {{Q}}u);
            {% for i in range(Q) -%}
                pdf.get({{i}}) = pop[offset + {{i}}u];
            {% endfor -%}
        }
    }

    __global__ void kernel_set(
        cuda::FieldAccessor< {{dtype}} > pdf,
        const {{dtype}} * RESTRICT const pop )
    {
        pdf.set( blockIdx, threadIdx );
        if (pdf.isValidPosition()) {
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, {{Q}}u);
            {% for i in range(Q) -%}
                pdf.get({{i}}) = pop[{{i}}u];
            {% endfor -%}
        }
    }

    std::array<{{dtype}}, {{Q}}u> get(
        cuda::GPUField< {{dtype}} > const * pdf_field,
        Cell const & cell )
    {
        CellInterval ci ( cell, cell );
        thrust::device_vector< {{dtype}} > dev_data({{Q}}u, {{dtype}} {0});
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_get );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *pdf_field, ci ) );
        kernel.addParam( dev_data_ptr );
        kernel();
        std::array<{{dtype}}, {{Q}}u> pop;
        thrust::copy(dev_data.begin(), dev_data.end(), pop.data());
        return pop;
    }

    void set(
        cuda::GPUField< {{dtype}} > * pdf_field,
        std::array< {{dtype}}, {{Q}}u > const & pop,
        Cell const & cell )
    {
        thrust::device_vector< {{dtype}} > dev_data(pop.data(), pop.data() + {{Q}}u);
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        CellInterval ci ( cell, cell );
        auto kernel = cuda::make_kernel( kernel_set );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *pdf_field, ci ) );
        kernel.addParam( const_cast<const {{dtype}} *>(dev_data_ptr) );
        kernel();
    }

    void broadcast(
        cuda::GPUField< {{dtype}} > * pdf_field,
        std::array< {{dtype}}, {{Q}}u > const & pop )
    {
        CellInterval ci = pdf_field->xyzSizeWithGhostLayer();
        thrust::device_vector< {{dtype}} > dev_data(pop.data(), pop.data() + {{Q}}u);
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_set );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *pdf_field, ci ) );
        kernel.addParam( const_cast<const {{dtype}} *>(dev_data_ptr) );
        kernel();
   }

    std::vector< {{dtype}} > get(
        cuda::GPUField< {{dtype}} > const * pdf_field,
        CellInterval const & ci )
    {
        thrust::device_vector< {{dtype}} > dev_data(ci.numCells() * {{Q}}u);
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_get_interval );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *pdf_field, ci ) );
        kernel.addParam( dev_data_ptr );
        kernel();
        std::vector< {{dtype}} > out(ci.numCells() * {{Q}}u);
        thrust::copy(dev_data.begin(), dev_data.end(), out.data());
        return out;
    }

    void set(
        cuda::GPUField< {{dtype}} > * pdf_field,
        std::vector< {{dtype}} > const & values,
        CellInterval const & ci )
    {
        thrust::device_vector< {{dtype}} > dev_data(values.begin(), values.end());
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_set_interval );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *pdf_field, ci ) );
        kernel.addParam( const_cast<const {{dtype}} *>(dev_data_ptr) );
        kernel();
    }
} // namespace Population

namespace Vector
{
    __global__ void kernel_get_interval(
        cuda::FieldAccessor< {{dtype}} > vec,
        {{dtype}} * const out )
    {
        vec.set( blockIdx, threadIdx );
        if (vec.isValidPosition()) {
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, {{D}}u);
            {% for i in range(D) -%}
                out[offset + {{i}}u] = vec.get({{i}});
            {% endfor %}
        }
    }

    __global__ void kernel_get(
        cuda::FieldAccessor< {{dtype}} > vec,
        {{dtype}} * const out )
    {
        vec.set( blockIdx, threadIdx );
        if (vec.isValidPosition()) {
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, {{D}}u);
            {% for i in range(D) -%}
                out[{{i}}u] = vec.get({{i}});
            {% endfor %}
        }
    }

    __global__ void kernel_set_interval(
        cuda::FieldAccessor< {{dtype}} > vec,
        const {{dtype}} * RESTRICT const u )
    {
        vec.set( blockIdx, threadIdx );
        if (vec.isValidPosition()) {
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, {{D}}u);
            {% for i in range(D) -%}
                vec.get({{i}}) = u[offset + {{i}}u];
            {% endfor %}
        }
    }

    __global__ void kernel_set(
        cuda::FieldAccessor< {{dtype}} > vec,
        const {{dtype}} * RESTRICT const u )
    {
        vec.set( blockIdx, threadIdx );
        if (vec.isValidPosition()) {
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, {{D}}u);
            {% for i in range(D) -%}
                vec.get({{i}}) = u[{{i}}u];
            {% endfor %}
        }
    }

    __global__ void kernel_add_interval(
        cuda::FieldAccessor< {{dtype}} > vec,
        const {{dtype}} * RESTRICT const u )
    {
        vec.set( blockIdx, threadIdx );
        if (vec.isValidPosition()) {
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, {{D}}u);
            {% for i in range(D) -%}
                vec.get({{i}}) += u[offset + {{i}}u];
            {% endfor %}
        }
    }

    __global__ void kernel_add(
        cuda::FieldAccessor< {{dtype}} > vec,
        const {{dtype}} * RESTRICT const u )
    {
        vec.set( blockIdx, threadIdx );
        if (vec.isValidPosition()) {
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, {{D}}u);
            {% for i in range(D) -%}
                vec.get({{i}}) += u[{{i}}u];
            {% endfor %}
        }
    }

    Vector{{D}}< {{dtype}} > get(
        cuda::GPUField< {{dtype}} > const * vec_field,
        Cell const & cell)
    {
        CellInterval ci ( cell, cell );
        thrust::device_vector< {{dtype}} > dev_data({{D}}u);
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_get );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *vec_field, ci ) );
        kernel.addParam( dev_data_ptr );
        kernel();
        Vector{{D}}< {{dtype}} > vec;
        thrust::copy(dev_data.begin(), dev_data.end(), vec.data());
        return vec;
    }

    void set(
        cuda::GPUField< {{dtype}} > * vec_field,
        Vector{{D}}< {{dtype}} > const & vec,
        Cell const & cell )
    {
        CellInterval ci ( cell, cell );
        thrust::device_vector< {{dtype}} > dev_data(vec.data(), vec.data() + {{D}}u);
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_set );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *vec_field, ci ) );
        kernel.addParam( const_cast<const {{dtype}} *>(dev_data_ptr) );
        kernel();
    }

    void add(
        cuda::GPUField< {{dtype}} > * vec_field,
        Vector{{D}}< {{dtype}} > const & vec,
        Cell const &cell )
    {
        CellInterval ci ( cell, cell );
        thrust::device_vector< {{dtype}} > dev_data(vec.data(), vec.data() + {{D}}u);
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_add );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *vec_field, ci ) );
        kernel.addParam( const_cast<const {{dtype}} *>(dev_data_ptr) );
        kernel();
    }

    void broadcast(
        cuda::GPUField< {{dtype}} > * vec_field,
        Vector{{D}}< {{dtype}} > const & vec )
    {
        CellInterval ci = vec_field->xyzSizeWithGhostLayer();
        thrust::device_vector< {{dtype}} > dev_data(vec.data(), vec.data() + {{D}}u);
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_set );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *vec_field, ci ) );
        kernel.addParam( const_cast<const {{dtype}} *>(dev_data_ptr) );
        kernel();
   }

    void add_to_all(
        cuda::GPUField< {{dtype}} > * vec_field,
        Vector{{D}}< {{dtype}} > const & vec )
    {
        CellInterval ci = vec_field->xyzSizeWithGhostLayer();
        thrust::device_vector< {{dtype}} > dev_data(vec.data(), vec.data() + {{D}}u);
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_add );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *vec_field, ci ) );
        kernel.addParam( const_cast<const {{dtype}} *>(dev_data_ptr) );
        kernel();
    }

    std::vector< {{dtype}} > get(
        cuda::GPUField< {{dtype}} > const * vec_field,
        CellInterval const & ci)
    {
        thrust::device_vector< {{dtype}} > dev_data(ci.numCells() * {{D}}u);
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_get_interval );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *vec_field, ci ) );
        kernel.addParam( dev_data_ptr );
        kernel();
        std::vector< {{dtype}} > out(ci.numCells() * {{D}}u);
        thrust::copy(dev_data.begin(), dev_data.end(), out.data());
        return out;
    }

    void set(
        cuda::GPUField< {{dtype}} > * vec_field,
        std::vector< {{dtype}} > const & values,
        CellInterval const & ci )
    {
        thrust::device_vector< {{dtype}} > dev_data(values.begin(), values.end());
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_set_interval );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *vec_field, ci ) );
        kernel.addParam( const_cast<const {{dtype}} *>(dev_data_ptr) );
        kernel();
    }
} // namespace Vector

namespace Equilibrium
{
    __device__ void kernel_set_device(
        cuda::FieldAccessor< {{dtype}} > pdf,
        const {{dtype}} * RESTRICT const u,
        {{dtype}} rho )
    {
        {%if not compressible %}
        rho -= {{dtype}}(1.0);
        {%endif %}

        {% for eqTerm in equilibrium -%}
            pdf.get({{loop.index0 }}) = {{eqTerm}};
        {% endfor -%}
    }
} // namespace Equilibrium

namespace Density
{
    __global__ void kernel_get(
        cuda::FieldAccessor< {{dtype}} > pdf,
        {{dtype}} * RESTRICT const out )
    {
        pdf.set( blockIdx, threadIdx );
        if (pdf.isValidPosition()) {
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, uint(1u));
            {% for i in range(Q) -%}
                const {{dtype}} f_{{i}} = pdf.get({{i}});
            {% endfor -%}
            {{density_getters | indent(12)}}
            out[offset] = rho;
        }
    }

    __global__ void kernel_set(
        cuda::FieldAccessor< {{dtype}} > pdf,
        const {{dtype}} * RESTRICT const rho_in )
    {
        pdf.set( blockIdx, threadIdx );
        if (pdf.isValidPosition()) {
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, uint(1u));
            {% for i in range(Q) -%}
                const {{dtype}} f_{{i}} = pdf.get({{i}});
            {% endfor -%}
            {{unshifted_momentum_density_getter | indent(12)}}

            // calculate current velocity (before density change)
            const {{dtype}} conversion = {{dtype}}(1) / rho;
            const {{dtype}} u_old[{{D}}] = { {% for i in range(D) %}momdensity_{{i}} * conversion{% if not loop.last %}, {% endif %}{% endfor %} };

            Equilibrium::kernel_set_device(pdf, u_old, rho_in[offset] {%if not compressible %} + {{dtype}}(1) {%endif%});
        }
    }

    {{dtype}} get(
        cuda::GPUField< {{dtype}} > const * pdf_field,
        Cell const & cell )
    {
        CellInterval ci ( cell, cell );
        thrust::device_vector< {{dtype}} > dev_data(1u);
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_get );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *pdf_field, ci ) );
        kernel.addParam( dev_data_ptr );
        kernel();
        {{dtype}} rho = dev_data[0u];
        return rho;
    }

    void set(
        cuda::GPUField< {{dtype}} > * pdf_field,
        const {{dtype}} rho,
        Cell const & cell )
    {
        CellInterval ci ( cell, cell );
        thrust::device_vector< {{dtype}} > dev_data(1u, rho);
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_set );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *pdf_field, ci ) );
        kernel.addParam( const_cast<const {{dtype}} *>(dev_data_ptr) );
        kernel();
    }

    std::vector< {{dtype}} > get(
        cuda::GPUField< {{dtype}} > const * pdf_field,
        CellInterval const & ci )
    {
        thrust::device_vector< {{dtype}} > dev_data(ci.numCells());
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_get );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *pdf_field, ci ) );
        kernel.addParam( dev_data_ptr );
        kernel();
        std::vector< {{dtype}} > out(ci.numCells());
        thrust::copy(dev_data.begin(), dev_data.end(), out.begin());
        return out;
    }

    void set(
        cuda::GPUField< {{dtype}} > * pdf_field,
        std::vector< {{dtype}} > const & values,
        CellInterval const & ci )
    {
        thrust::device_vector< {{dtype}} > dev_data(values.begin(), values.end());
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_set );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *pdf_field, ci ) );
        kernel.addParam( const_cast<const {{dtype}} *>(dev_data_ptr) );
        kernel();
    }
} // namespace Density

namespace Velocity
{
    __global__ void kernel_set(
        cuda::FieldAccessor< {{dtype}} > pdf,
        cuda::FieldAccessor< {{dtype}} > force,
        const {{dtype}} * RESTRICT const u_in )
    {
        pdf.set( blockIdx, threadIdx );
        force.set( blockIdx, threadIdx );
        if (pdf.isValidPosition()) {
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, uint({{D}}u));
            const uint_t bufsize = {{D}}u;
            const {{dtype}} * RESTRICT const u = u_in + bufsize * offset;
            {% for i in range(Q) -%}
                const {{dtype}} f_{{i}} = pdf.get({{i}});
            {% endfor -%}
            {{density_getters | indent(8)}}
            {{density_velocity_setter_macroscopic_values | substitute_force_getter_cu | indent(8)}}
            {{dtype}} u_new[{{D}}] = { {% for i in range(D) %}u_{{i}}{% if not loop.last %}, {% endif %}{% endfor %} };

            Equilibrium::kernel_set_device(pdf, u_new, rho {%if not compressible %} + {{dtype}}(1) {%endif%});
        }
    }

    void set(
        cuda::GPUField< {{dtype}} > * pdf_field,
        cuda::GPUField< {{dtype}} > * force_field,
        Vector{{D}}< {{dtype}} > const & u,
        Cell const & cell )
    {
        CellInterval ci ( cell, cell );
        thrust::device_vector< {{dtype}} > dev_data(u.data(), u.data() + {{D}}u);
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_set );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *pdf_field, ci ) );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *force_field, ci ) );
        kernel.addParam( const_cast<const {{dtype}} *>(dev_data_ptr) );
        kernel();
    }
} // namespace Velocity

namespace MomentumDensity
{
    __global__ void kernel_sum(
        cuda::FieldAccessor< {{dtype}} > pdf,
        cuda::FieldAccessor< {{dtype}} > force,
        {{dtype}} * RESTRICT const out )
    {
        pdf.set( blockIdx, threadIdx );
        force.set( blockIdx, threadIdx );
        if (pdf.isValidPosition()) {
            const uint bufsize = {{D}}u;
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, bufsize);
            {% for i in range(Q) -%}
                const {{dtype}} f_{{i}} = pdf.get({{i}});
            {% endfor -%}
            {{momentum_density_getter | substitute_force_getter_cu | indent(8) }}
            {% for i in range(D) -%}
                out[bufsize * offset + {{i}}u] += md_{{i}};
            {% endfor %}
        }
    }

    Vector{{D}}< {{dtype}} > reduce(
        cuda::GPUField< {{dtype}} > const * pdf_field,
        cuda::GPUField< {{dtype}} > const * force_field )
    {
        thrust::device_vector< {{dtype}} > dev_data({{D}}u, {{dtype}} {0});
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        WALBERLA_FOR_ALL_CELLS_XYZ(pdf_field, {
            Cell cell(x, y, z);
            CellInterval ci ( cell, cell );
            auto kernel = cuda::make_kernel( kernel_sum );
            kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *pdf_field, ci ) );
            kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *force_field, ci ) );
            kernel.addParam( dev_data_ptr );
            kernel();
        });
        Vector{{D}}< {{dtype}} > mom({{dtype}} {0});
        thrust::copy(dev_data.begin(), dev_data.begin() + {{D}}u, mom.data());
        return mom;
    }
} // namespace MomentumDensity

namespace PressureTensor
{
    __global__ void kernel_get(
        cuda::FieldAccessor< {{dtype}} > pdf,
        {{dtype}} * RESTRICT const out )
    {
        pdf.set( blockIdx, threadIdx );
        if (pdf.isValidPosition()) {
            const uint bufsize = {{D**2}}u;
            const uint offset = getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, bufsize);
            {% for i in range(Q) -%}
                const {{dtype}} f_{{i}} = pdf.get({{i}});
            {% endfor -%}
            {{second_momentum_getter | indent(12) }}
            {% for i in range(D) -%}
                {% for j in range(D) -%}
                    out[bufsize * offset + {{i*D+j}}u] = p_{{i*D+j}};
                {% endfor %}
            {% endfor %}
        }
    }

    Matrix{{D}}< {{dtype}} > get(
        cuda::GPUField< {{dtype}} > const * pdf_field,
        Cell const & cell )
    {
        CellInterval ci ( cell, cell );
        thrust::device_vector< {{dtype}} > dev_data({{D**2}}u);
        auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
        auto kernel = cuda::make_kernel( kernel_get );
        kernel.addFieldIndexingParam( cuda::FieldIndexing< {{dtype}} >::interval( *pdf_field, ci ) );
        kernel.addParam( dev_data_ptr );
        kernel();
        Matrix{{D}}< {{dtype}} > out;
        thrust::copy(dev_data.begin(), dev_data.begin() + {{D**2}}u, out.data());
        return out;
   }
} // namespace PressureTensor


} // namespace accessor
} // namespace {{namespace}}
} // namespace walberla
