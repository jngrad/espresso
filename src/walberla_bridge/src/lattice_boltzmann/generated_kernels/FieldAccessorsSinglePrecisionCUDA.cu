#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023-2024 The ESPResSo project
 * Copyright (C) 2020 The waLBerla project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

// kernel generated with pystencils v1.2, lbmpy v1.2,
// lbmpy_walberla/pystencils_walberla from waLBerla commit
// 065ce5f311850371a97ac4766f47dbb5ca8424ba

/**
 * @file
 * Lattice field accessors.
 * Adapted from the waLBerla source file
 * https://i10git.cs.fau.de/walberla/walberla/-/blob/a16141524c58ab88386e2a0f8fdd7c63c5edd704/python/lbmpy_walberla/templates/LatticeModel.tmpl.h
 */

#include <core/DataTypes.h>
#include <core/cell/Cell.h>
#include <core/cell/CellInterval.h>
#include <core/math/Matrix3.h>
#include <core/math/Vector3.h>

#include <field/iterators/IteratorMacros.h>

#include <cuda/FieldAccessor.h>
#include <cuda/FieldIndexing.h>
#include <cuda/GPUField.h>
#include <cuda/Kernel.h>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

#include <array>
#include <tuple>
#include <vector>

#ifdef __GNUC__
#define RESTRICT __restrict__
#elif _MSC_VER
#define RESTRICT __restrict
#else
#define RESTRICT
#endif

__device__ inline uint get_num_threads(uint3 gridDim, uint3 blockDim) {
  return gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y *
         blockDim.z;
}

__device__ inline uint getLinearIndexXYZF(uint3 blockIdx, uint3 threadIdx,
                                          uint3 gridDim, uint3 blockDim) {
  auto const x = threadIdx.x;
  auto const y = blockIdx.x;
  auto const z = blockIdx.y;
  auto const f = blockIdx.z;
  auto const xSize = blockDim.x;
  auto const ySize = gridDim.x;
  auto const zSize = gridDim.y;
  return x + y * xSize + z * xSize * ySize + f * xSize * ySize * zSize;
}

__device__ inline uint getLinearIndexFZYX(uint3 blockIdx, uint3 threadIdx,
                                          uint3 gridDim, uint3 blockDim,
                                          uint fOffset) {
  auto const x = threadIdx.x;
  auto const y = blockIdx.x;
  auto const z = blockIdx.y;
  auto const f = blockIdx.z;
  auto const ySize = gridDim.x;
  auto const zSize = gridDim.y;
  auto const fSize = fOffset;
  return f + z * fSize + y * fSize * zSize + x * fSize * zSize * ySize;
}

namespace walberla {
namespace lbm {
namespace accessor {

namespace Population {
__global__ void kernel_get_interval(cuda::FieldAccessor<float> pdf,
                                    float *RESTRICT const pop) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, 19u);
    pop[offset + 0u] = pdf.get(0);
    pop[offset + 1u] = pdf.get(1);
    pop[offset + 2u] = pdf.get(2);
    pop[offset + 3u] = pdf.get(3);
    pop[offset + 4u] = pdf.get(4);
    pop[offset + 5u] = pdf.get(5);
    pop[offset + 6u] = pdf.get(6);
    pop[offset + 7u] = pdf.get(7);
    pop[offset + 8u] = pdf.get(8);
    pop[offset + 9u] = pdf.get(9);
    pop[offset + 10u] = pdf.get(10);
    pop[offset + 11u] = pdf.get(11);
    pop[offset + 12u] = pdf.get(12);
    pop[offset + 13u] = pdf.get(13);
    pop[offset + 14u] = pdf.get(14);
    pop[offset + 15u] = pdf.get(15);
    pop[offset + 16u] = pdf.get(16);
    pop[offset + 17u] = pdf.get(17);
    pop[offset + 18u] = pdf.get(18);
  }
}

__global__ void kernel_get(cuda::FieldAccessor<float> pdf,
                           float *RESTRICT const pop) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, 19u);
    pop[0u] = pdf.get(0);
    pop[1u] = pdf.get(1);
    pop[2u] = pdf.get(2);
    pop[3u] = pdf.get(3);
    pop[4u] = pdf.get(4);
    pop[5u] = pdf.get(5);
    pop[6u] = pdf.get(6);
    pop[7u] = pdf.get(7);
    pop[8u] = pdf.get(8);
    pop[9u] = pdf.get(9);
    pop[10u] = pdf.get(10);
    pop[11u] = pdf.get(11);
    pop[12u] = pdf.get(12);
    pop[13u] = pdf.get(13);
    pop[14u] = pdf.get(14);
    pop[15u] = pdf.get(15);
    pop[16u] = pdf.get(16);
    pop[17u] = pdf.get(17);
    pop[18u] = pdf.get(18);
  }
}

__global__ void kernel_set_interval(cuda::FieldAccessor<float> pdf,
                                    const float *RESTRICT const pop) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, 19u);
    pdf.get(0) = pop[offset + 0u];
    pdf.get(1) = pop[offset + 1u];
    pdf.get(2) = pop[offset + 2u];
    pdf.get(3) = pop[offset + 3u];
    pdf.get(4) = pop[offset + 4u];
    pdf.get(5) = pop[offset + 5u];
    pdf.get(6) = pop[offset + 6u];
    pdf.get(7) = pop[offset + 7u];
    pdf.get(8) = pop[offset + 8u];
    pdf.get(9) = pop[offset + 9u];
    pdf.get(10) = pop[offset + 10u];
    pdf.get(11) = pop[offset + 11u];
    pdf.get(12) = pop[offset + 12u];
    pdf.get(13) = pop[offset + 13u];
    pdf.get(14) = pop[offset + 14u];
    pdf.get(15) = pop[offset + 15u];
    pdf.get(16) = pop[offset + 16u];
    pdf.get(17) = pop[offset + 17u];
    pdf.get(18) = pop[offset + 18u];
  }
}

__global__ void kernel_set(cuda::FieldAccessor<float> pdf,
                           const float *RESTRICT const pop) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, 19u);
    pdf.get(0) = pop[0u];
    pdf.get(1) = pop[1u];
    pdf.get(2) = pop[2u];
    pdf.get(3) = pop[3u];
    pdf.get(4) = pop[4u];
    pdf.get(5) = pop[5u];
    pdf.get(6) = pop[6u];
    pdf.get(7) = pop[7u];
    pdf.get(8) = pop[8u];
    pdf.get(9) = pop[9u];
    pdf.get(10) = pop[10u];
    pdf.get(11) = pop[11u];
    pdf.get(12) = pop[12u];
    pdf.get(13) = pop[13u];
    pdf.get(14) = pop[14u];
    pdf.get(15) = pop[15u];
    pdf.get(16) = pop[16u];
    pdf.get(17) = pop[17u];
    pdf.get(18) = pop[18u];
  }
}

std::array<float, 19u> get(cuda::GPUField<float> const *pdf_field,
                           Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<float> dev_data(19u, float{0});
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::array<float, 19u> pop;
  thrust::copy(dev_data.begin(), dev_data.end(), pop.data());
  return pop;
}

void set(cuda::GPUField<float> *pdf_field, std::array<float, 19u> const &pop,
         Cell const &cell) {
  thrust::device_vector<float> dev_data(pop.data(), pop.data() + 19u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  CellInterval ci(cell, cell);
  auto kernel = cuda::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const float *>(dev_data_ptr));
  kernel();
}

void broadcast(cuda::GPUField<float> *pdf_field,
               std::array<float, 19u> const &pop) {
  CellInterval ci = pdf_field->xyzSizeWithGhostLayer();
  thrust::device_vector<float> dev_data(pop.data(), pop.data() + 19u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const float *>(dev_data_ptr));
  kernel();
}

std::vector<float> get(cuda::GPUField<float> const *pdf_field,
                       CellInterval const &ci) {
  thrust::device_vector<float> dev_data(ci.numCells() * 19u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_get_interval);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::vector<float> out(ci.numCells() * 19u);
  thrust::copy(dev_data.begin(), dev_data.end(), out.data());
  return out;
}

void set(cuda::GPUField<float> *pdf_field, std::vector<float> const &values,
         CellInterval const &ci) {
  thrust::device_vector<float> dev_data(values.begin(), values.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_set_interval);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const float *>(dev_data_ptr));
  kernel();
}
} // namespace Population

namespace Vector {
__global__ void kernel_get_interval(cuda::FieldAccessor<float> vec,
                                    float *const out) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, 3u);
    out[offset + 0u] = vec.get(0);
    out[offset + 1u] = vec.get(1);
    out[offset + 2u] = vec.get(2);
  }
}

__global__ void kernel_get(cuda::FieldAccessor<float> vec, float *const out) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, 3u);
    out[0u] = vec.get(0);
    out[1u] = vec.get(1);
    out[2u] = vec.get(2);
  }
}

__global__ void kernel_set_interval(cuda::FieldAccessor<float> vec,
                                    const float *RESTRICT const u) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, 3u);
    vec.get(0) = u[offset + 0u];
    vec.get(1) = u[offset + 1u];
    vec.get(2) = u[offset + 2u];
  }
}

__global__ void kernel_set(cuda::FieldAccessor<float> vec,
                           const float *RESTRICT const u) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, 3u);
    vec.get(0) = u[0u];
    vec.get(1) = u[1u];
    vec.get(2) = u[2u];
  }
}

__global__ void kernel_add_interval(cuda::FieldAccessor<float> vec,
                                    const float *RESTRICT const u) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, 3u);
    vec.get(0) += u[offset + 0u];
    vec.get(1) += u[offset + 1u];
    vec.get(2) += u[offset + 2u];
  }
}

__global__ void kernel_add(cuda::FieldAccessor<float> vec,
                           const float *RESTRICT const u) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, 3u);
    vec.get(0) += u[0u];
    vec.get(1) += u[1u];
    vec.get(2) += u[2u];
  }
}

__global__ void kernel_get_part_coupling(cuda::FieldAccessor<float> vec,
                                         float const *RESTRICT const pos,
                                         float *RESTRICT const vel, uint n_part,
                                         uint gl) {

  unsigned int part_index = blockIdx.y * gridDim.x * blockDim.x +
                            blockDim.x * blockIdx.x + threadIdx.x;

  vec.set({0u, 0u, 0u}, {0u, 0u, 0u});
  if (vec.isValidPosition() and part_index < n_part) {
    auto const array_offset = part_index * 3u;
    int corner[3];
    float distance[3];
    for (unsigned int dim = 0u; dim < 3u; ++dim) {
      auto const fractional_index = pos[array_offset + dim] - 0.5f;
      auto const nmp = floorf(fractional_index);
      distance[dim] = fractional_index - nmp - 0.5f;
      corner[dim] = __float2int_rn(nmp) + gl;
    }
    float w_x[2] = {0.5f - distance[0], 0.5f + distance[0]};
    float w_y[2] = {0.5f - distance[1], 0.5f + distance[1]};
    float w_z[2] = {0.5f - distance[2], 0.5f + distance[2]};
    for (int i = 0; i < 2; i++) {
      auto const cx = corner[0] + i;
      auto const wx = w_x[static_cast<unsigned>(i)];
      for (int j = 0; j < 2; j++) {
        auto const cy = corner[1] + j;
        auto const wxy = wx * w_y[static_cast<unsigned>(j)];
        for (int k = 0; k < 2; k++) {
          auto const cz = corner[2] + k;
          auto const weight = wxy * w_z[static_cast<unsigned>(k)];
          vel[array_offset + 0u] += weight * vec.getNeighbor(cx, cy, cz, 0);
          vel[array_offset + 1u] += weight * vec.getNeighbor(cx, cy, cz, 1);
          vel[array_offset + 2u] += weight * vec.getNeighbor(cx, cy, cz, 2);
        }
      }
    }
  }
}

__global__ void kernel_set_part_coupling(cuda::FieldAccessor<float> vec,
                                         float const *RESTRICT const pos,
                                         float const *RESTRICT const forces,
                                         uint n_part, uint gl) {

  unsigned int part_index = blockIdx.y * gridDim.x * blockDim.x +
                            blockDim.x * blockIdx.x + threadIdx.x;

  vec.set({0u, 0u, 0u}, {0u, 0u, 0u});
  if (vec.isValidPosition() and part_index < n_part) {
    auto const array_offset = part_index * 3u;
    int corner[3];
    float distance[3];
    for (unsigned int dim = 0u; dim < 3u; ++dim) {
      auto const fractional_index = pos[array_offset + dim] - 0.5f;
      auto const nmp = floorf(fractional_index);
      distance[dim] = fractional_index - nmp - 0.5f;
      corner[dim] = __float2int_rn(nmp) + static_cast<int>(gl);
    }
    float w_x[2] = {0.5f - distance[0], 0.5f + distance[0]};
    float w_y[2] = {0.5f - distance[1], 0.5f + distance[1]};
    float w_z[2] = {0.5f - distance[2], 0.5f + distance[2]};
    for (int i = 0; i < 2; i++) {
      auto const cx = corner[0] + i;
      auto const wx = w_x[static_cast<unsigned>(i)];
      for (int j = 0; j < 2; j++) {
        auto const cy = corner[1] + j;
        auto const wxy = wx * w_y[static_cast<unsigned>(j)];
        for (int k = 0; k < 2; k++) {
          auto const cz = corner[2] + k;
          auto const weight = wxy * w_z[static_cast<unsigned>(k)];
          atomicAdd(&vec.getNeighbor(cx, cy, cz, 0),
                    weight * forces[array_offset + 0u]);
          atomicAdd(&vec.getNeighbor(cx, cy, cz, 1),
                    weight * forces[array_offset + 1u]);
          atomicAdd(&vec.getNeighbor(cx, cy, cz, 2),
                    weight * forces[array_offset + 2u]);
        }
      }
    }
  }
}

inline dim3 calculate_dim_grid(unsigned const threads_x,
                               unsigned const blocks_per_grid_y,
                               unsigned const threads_per_block) {
  assert(threads_x >= 1);
  assert(blocks_per_grid_y >= 1);
  assert(threads_per_block >= 1);
  auto const threads_y = threads_per_block * blocks_per_grid_y;
  auto const blocks_per_grid_x = (threads_x + threads_y - 1) / threads_y;
  return make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);
}

std::vector<float> get_part_coupling(cuda::GPUField<float> const *vec_field,
                                     std::vector<float> const &pos, uint gl) {
  thrust::device_vector<float> dev_pos(pos.begin(), pos.end());
  thrust::device_vector<float> dev_vel(pos.size());
  auto const dev_pos_ptr = thrust::raw_pointer_cast(dev_pos.data());
  auto const dev_vel_ptr = thrust::raw_pointer_cast(dev_vel.data());

  auto const threads_per_block = 64u;
  auto const n_part = pos.size() / 3ul;
  dim3 dim_grid =
      calculate_dim_grid(static_cast<unsigned>(n_part), 4u, threads_per_block);
  kernel_get_part_coupling<<<dim_grid, threads_per_block, 0u, nullptr>>>(
      cuda::FieldIndexing<float>::withGhostLayerXYZ(*vec_field, gl).gpuAccess(),
      dev_pos_ptr, dev_vel_ptr, static_cast<uint>(pos.size() / 3ul), gl);

  std::vector<float> out(pos.size());
  thrust::copy(dev_vel.begin(), dev_vel.end(), out.data());
  return out;
}

void set_part_coupling(cuda::GPUField<float> const *vec_field,
                       std::vector<float> const &pos,
                       std::vector<float> const &forces, uint gl) {
  thrust::device_vector<float> dev_pos(pos.begin(), pos.end());
  thrust::device_vector<float> dev_for(forces.begin(), forces.end());
  auto const dev_pos_ptr = thrust::raw_pointer_cast(dev_pos.data());
  auto const dev_for_ptr = thrust::raw_pointer_cast(dev_for.data());

  auto const threads_per_block = 64u;
  auto const n_part = pos.size() / 3ul;
  dim3 dim_grid =
      calculate_dim_grid(static_cast<unsigned>(n_part), 4u, threads_per_block);
  kernel_set_part_coupling<<<dim_grid, threads_per_block, 0u, nullptr>>>(
      cuda::FieldIndexing<float>::withGhostLayerXYZ(*vec_field, gl).gpuAccess(),
      dev_pos_ptr, dev_for_ptr, static_cast<uint>(pos.size() / 3ul), gl);
}

Vector3<float> get(cuda::GPUField<float> const *vec_field, Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<float> dev_data(3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*vec_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  Vector3<float> vec;
  thrust::copy(dev_data.begin(), dev_data.end(), vec.data());
  return vec;
}

void set(cuda::GPUField<float> *vec_field, Vector3<float> const &vec,
         Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<float> dev_data(vec.data(), vec.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const float *>(dev_data_ptr));
  kernel();
}

void add(cuda::GPUField<float> *vec_field, Vector3<float> const &vec,
         Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<float> dev_data(vec.data(), vec.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_add);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const float *>(dev_data_ptr));
  kernel();
}

__global__ void kernel_add_at(cuda::FieldAccessor<float> vec,
                              const float *RESTRICT const v,
                              const int *RESTRICT const c, uint size) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    auto const x = threadIdx.x;
    auto const y = blockIdx.x;
    auto const z = blockIdx.y;
    for (uint offset = 0u; offset < size; offset += 3u) {
      auto const x_ = c[offset + 0u];
      auto const y_ = c[offset + 1u];
      auto const z_ = c[offset + 2u];
      if (x == x_ and y == y_ and z == z_) {
        vec.get(0) += v[offset + 0u];
        vec.get(1) += v[offset + 1u];
        vec.get(2) += v[offset + 2u];
      }
    }
  }
}

__global__ void kernel_get_at(cuda::FieldAccessor<float> vec,
                              float *RESTRICT const v,
                              const int *RESTRICT const c, uint size) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    auto const x = threadIdx.x;
    auto const y = blockIdx.x;
    auto const z = blockIdx.y;
    for (uint offset = 0u; offset < size; offset += 3u) {
      auto const x_ = c[offset + 0u];
      auto const y_ = c[offset + 1u];
      auto const z_ = c[offset + 2u];
      if (x == x_ and y == y_ and z == z_) {
        v[offset + 0u] = vec.get(0);
        v[offset + 1u] = vec.get(1);
        v[offset + 2u] = vec.get(2);
      }
    }
  }
}

void add_at(cuda::GPUField<float> *vec_field, std::vector<float> const &vecs,
            std::vector<cell_idx_t> const &cells) {
  CellInterval ci = vec_field->xyzSizeWithGhostLayer();
  thrust::device_vector<float> dev_data(vecs.begin(), vecs.end());
  thrust::device_vector<cell_idx_t> dev_cell(cells.begin(), cells.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto const dev_cell_ptr = thrust::raw_pointer_cast(dev_cell.data());
  auto kernel = cuda::make_kernel(kernel_add_at);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const float *>(dev_data_ptr));
  kernel.addParam(const_cast<const int *>(dev_cell_ptr));
  kernel.addParam(static_cast<unsigned int>(cells.size()));
  kernel();
}

std::vector<float> get_at(cuda::GPUField<float> *vec_field,
                          std::vector<cell_idx_t> const &cells) {
  CellInterval ci = vec_field->xyzSizeWithGhostLayer();
  thrust::device_vector<float> dev_data(cells.size());
  thrust::device_vector<cell_idx_t> dev_cell(cells.begin(), cells.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto const dev_cell_ptr = thrust::raw_pointer_cast(dev_cell.data());
  auto kernel = cuda::make_kernel(kernel_get_at);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*vec_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel.addParam(const_cast<const int *>(dev_cell_ptr));
  kernel.addParam(static_cast<unsigned int>(cells.size()));
  kernel();
  std::vector<float> out(cells.size());
  thrust::copy(dev_data.begin(), dev_data.end(), out.data());
  return out;
}

void broadcast(cuda::GPUField<float> *vec_field, Vector3<float> const &vec) {
  CellInterval ci = vec_field->xyzSizeWithGhostLayer();
  thrust::device_vector<float> dev_data(vec.data(), vec.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const float *>(dev_data_ptr));
  kernel();
}

void add_to_all(cuda::GPUField<float> *vec_field, Vector3<float> const &vec) {
  CellInterval ci = vec_field->xyzSizeWithGhostLayer();
  thrust::device_vector<float> dev_data(vec.data(), vec.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_add);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const float *>(dev_data_ptr));
  kernel();
}

std::vector<float> get(cuda::GPUField<float> const *vec_field,
                       CellInterval const &ci) {
  thrust::device_vector<float> dev_data(ci.numCells() * 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_get_interval);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*vec_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::vector<float> out(ci.numCells() * 3u);
  thrust::copy(dev_data.begin(), dev_data.end(), out.data());
  return out;
}

void set(cuda::GPUField<float> *vec_field, std::vector<float> const &values,
         CellInterval const &ci) {
  thrust::device_vector<float> dev_data(values.begin(), values.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_set_interval);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const float *>(dev_data_ptr));
  kernel();
}
} // namespace Vector

namespace Equilibrium {
__device__ void kernel_set_device(cuda::FieldAccessor<float> pdf,
                                  const float *RESTRICT const u, float rho) {

  pdf.get(0) = rho * -0.33333333333333331f * (u[0] * u[0]) +
               rho * -0.33333333333333331f * (u[1] * u[1]) +
               rho * -0.33333333333333331f * (u[2] * u[2]) +
               rho * 0.33333333333333331f;
  pdf.get(1) = rho * -0.16666666666666666f * (u[0] * u[0]) +
               rho * -0.16666666666666666f * (u[2] * u[2]) +
               rho * 0.055555555555555552f + rho * 0.16666666666666666f * u[1] +
               rho * 0.16666666666666666f * (u[1] * u[1]);
  pdf.get(2) = rho * -0.16666666666666666f * u[1] +
               rho * -0.16666666666666666f * (u[0] * u[0]) +
               rho * -0.16666666666666666f * (u[2] * u[2]) +
               rho * 0.055555555555555552f +
               rho * 0.16666666666666666f * (u[1] * u[1]);
  pdf.get(3) = rho * -0.16666666666666666f * u[0] +
               rho * -0.16666666666666666f * (u[1] * u[1]) +
               rho * -0.16666666666666666f * (u[2] * u[2]) +
               rho * 0.055555555555555552f +
               rho * 0.16666666666666666f * (u[0] * u[0]);
  pdf.get(4) = rho * -0.16666666666666666f * (u[1] * u[1]) +
               rho * -0.16666666666666666f * (u[2] * u[2]) +
               rho * 0.055555555555555552f + rho * 0.16666666666666666f * u[0] +
               rho * 0.16666666666666666f * (u[0] * u[0]);
  pdf.get(5) = rho * -0.16666666666666666f * (u[0] * u[0]) +
               rho * -0.16666666666666666f * (u[1] * u[1]) +
               rho * 0.055555555555555552f + rho * 0.16666666666666666f * u[2] +
               rho * 0.16666666666666666f * (u[2] * u[2]);
  pdf.get(6) = rho * -0.16666666666666666f * u[2] +
               rho * -0.16666666666666666f * (u[0] * u[0]) +
               rho * -0.16666666666666666f * (u[1] * u[1]) +
               rho * 0.055555555555555552f +
               rho * 0.16666666666666666f * (u[2] * u[2]);
  pdf.get(7) = rho * -0.083333333333333329f * u[0] +
               rho * -0.25f * u[0] * u[1] + rho * 0.027777777777777776f +
               rho * 0.083333333333333329f * u[1] +
               rho * 0.083333333333333329f * (u[0] * u[0]) +
               rho * 0.083333333333333329f * (u[1] * u[1]);
  pdf.get(8) =
      rho * 0.027777777777777776f + rho * 0.083333333333333329f * u[0] +
      rho * 0.083333333333333329f * u[1] +
      rho * 0.083333333333333329f * (u[0] * u[0]) +
      rho * 0.083333333333333329f * (u[1] * u[1]) + rho * 0.25f * u[0] * u[1];
  pdf.get(9) =
      rho * -0.083333333333333329f * u[0] +
      rho * -0.083333333333333329f * u[1] + rho * 0.027777777777777776f +
      rho * 0.083333333333333329f * (u[0] * u[0]) +
      rho * 0.083333333333333329f * (u[1] * u[1]) + rho * 0.25f * u[0] * u[1];
  pdf.get(10) = rho * -0.083333333333333329f * u[1] +
                rho * -0.25f * u[0] * u[1] + rho * 0.027777777777777776f +
                rho * 0.083333333333333329f * u[0] +
                rho * 0.083333333333333329f * (u[0] * u[0]) +
                rho * 0.083333333333333329f * (u[1] * u[1]);
  pdf.get(11) =
      rho * 0.027777777777777776f + rho * 0.083333333333333329f * u[1] +
      rho * 0.083333333333333329f * u[2] +
      rho * 0.083333333333333329f * (u[1] * u[1]) +
      rho * 0.083333333333333329f * (u[2] * u[2]) + rho * 0.25f * u[1] * u[2];
  pdf.get(12) = rho * -0.083333333333333329f * u[1] +
                rho * -0.25f * u[1] * u[2] + rho * 0.027777777777777776f +
                rho * 0.083333333333333329f * u[2] +
                rho * 0.083333333333333329f * (u[1] * u[1]) +
                rho * 0.083333333333333329f * (u[2] * u[2]);
  pdf.get(13) = rho * -0.083333333333333329f * u[0] +
                rho * -0.25f * u[0] * u[2] + rho * 0.027777777777777776f +
                rho * 0.083333333333333329f * u[2] +
                rho * 0.083333333333333329f * (u[0] * u[0]) +
                rho * 0.083333333333333329f * (u[2] * u[2]);
  pdf.get(14) =
      rho * 0.027777777777777776f + rho * 0.083333333333333329f * u[0] +
      rho * 0.083333333333333329f * u[2] +
      rho * 0.083333333333333329f * (u[0] * u[0]) +
      rho * 0.083333333333333329f * (u[2] * u[2]) + rho * 0.25f * u[0] * u[2];
  pdf.get(15) = rho * -0.083333333333333329f * u[2] +
                rho * -0.25f * u[1] * u[2] + rho * 0.027777777777777776f +
                rho * 0.083333333333333329f * u[1] +
                rho * 0.083333333333333329f * (u[1] * u[1]) +
                rho * 0.083333333333333329f * (u[2] * u[2]);
  pdf.get(16) =
      rho * -0.083333333333333329f * u[1] +
      rho * -0.083333333333333329f * u[2] + rho * 0.027777777777777776f +
      rho * 0.083333333333333329f * (u[1] * u[1]) +
      rho * 0.083333333333333329f * (u[2] * u[2]) + rho * 0.25f * u[1] * u[2];
  pdf.get(17) =
      rho * -0.083333333333333329f * u[0] +
      rho * -0.083333333333333329f * u[2] + rho * 0.027777777777777776f +
      rho * 0.083333333333333329f * (u[0] * u[0]) +
      rho * 0.083333333333333329f * (u[2] * u[2]) + rho * 0.25f * u[0] * u[2];
  pdf.get(18) = rho * -0.083333333333333329f * u[2] +
                rho * -0.25f * u[0] * u[2] + rho * 0.027777777777777776f +
                rho * 0.083333333333333329f * u[0] +
                rho * 0.083333333333333329f * (u[0] * u[0]) +
                rho * 0.083333333333333329f * (u[2] * u[2]);
}
} // namespace Equilibrium

namespace Density {
__global__ void kernel_get(cuda::FieldAccessor<float> pdf,
                           float *RESTRICT const out) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, uint(1u));
    const float f_0 = pdf.get(0);
    const float f_1 = pdf.get(1);
    const float f_2 = pdf.get(2);
    const float f_3 = pdf.get(3);
    const float f_4 = pdf.get(4);
    const float f_5 = pdf.get(5);
    const float f_6 = pdf.get(6);
    const float f_7 = pdf.get(7);
    const float f_8 = pdf.get(8);
    const float f_9 = pdf.get(9);
    const float f_10 = pdf.get(10);
    const float f_11 = pdf.get(11);
    const float f_12 = pdf.get(12);
    const float f_13 = pdf.get(13);
    const float f_14 = pdf.get(14);
    const float f_15 = pdf.get(15);
    const float f_16 = pdf.get(16);
    const float f_17 = pdf.get(17);
    const float f_18 = pdf.get(18);
    const float vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const float vel1Term = f_1 + f_11 + f_15 + f_7;
    const float vel2Term = f_12 + f_13 + f_5;
    const float rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term +
                      vel1Term + vel2Term;
    out[offset] = rho;
  }
}

__global__ void kernel_set(cuda::FieldAccessor<float> pdf,
                           const float *RESTRICT const rho_in) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, uint(1u));
    const float f_0 = pdf.get(0);
    const float f_1 = pdf.get(1);
    const float f_2 = pdf.get(2);
    const float f_3 = pdf.get(3);
    const float f_4 = pdf.get(4);
    const float f_5 = pdf.get(5);
    const float f_6 = pdf.get(6);
    const float f_7 = pdf.get(7);
    const float f_8 = pdf.get(8);
    const float f_9 = pdf.get(9);
    const float f_10 = pdf.get(10);
    const float f_11 = pdf.get(11);
    const float f_12 = pdf.get(12);
    const float f_13 = pdf.get(13);
    const float f_14 = pdf.get(14);
    const float f_15 = pdf.get(15);
    const float f_16 = pdf.get(16);
    const float f_17 = pdf.get(17);
    const float f_18 = pdf.get(18);
    const float vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const float momdensity_0 = -f_13 - f_17 - f_3 - f_7 - f_9 + vel0Term;
    const float vel1Term = f_1 + f_11 + f_15 + f_7;
    const float momdensity_1 = -f_10 - f_12 - f_16 - f_2 + f_8 - f_9 + vel1Term;
    const float vel2Term = f_12 + f_13 + f_5;
    const float momdensity_2 =
        f_11 + f_14 - f_15 - f_16 - f_17 - f_18 - f_6 + vel2Term;
    const float rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term +
                      vel1Term + vel2Term;

    // calculate current velocity (before density change)
    const float conversion = float(1) / rho;
    const float u_old[3] = {momdensity_0 * conversion,
                            momdensity_1 * conversion,
                            momdensity_2 * conversion};

    Equilibrium::kernel_set_device(pdf, u_old, rho_in[offset]);
  }
}

float get(cuda::GPUField<float> const *pdf_field, Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<float> dev_data(1u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  float rho = dev_data[0u];
  return rho;
}

void set(cuda::GPUField<float> *pdf_field, const float rho, Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<float> dev_data(1u, rho);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const float *>(dev_data_ptr));
  kernel();
}

std::vector<float> get(cuda::GPUField<float> const *pdf_field,
                       CellInterval const &ci) {
  thrust::device_vector<float> dev_data(ci.numCells());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::vector<float> out(ci.numCells());
  thrust::copy(dev_data.begin(), dev_data.end(), out.begin());
  return out;
}

void set(cuda::GPUField<float> *pdf_field, std::vector<float> const &values,
         CellInterval const &ci) {
  thrust::device_vector<float> dev_data(values.begin(), values.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const float *>(dev_data_ptr));
  kernel();
}
} // namespace Density

namespace Velocity {
__global__ void kernel_set(cuda::FieldAccessor<float> pdf,
                           cuda::FieldAccessor<float> force,
                           const float *RESTRICT const u_in) {
  pdf.set(blockIdx, threadIdx);
  force.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, uint(3u));
    const uint_t bufsize = 3u;
    const float *RESTRICT const u = u_in + bufsize * offset;
    const float f_0 = pdf.get(0);
    const float f_1 = pdf.get(1);
    const float f_2 = pdf.get(2);
    const float f_3 = pdf.get(3);
    const float f_4 = pdf.get(4);
    const float f_5 = pdf.get(5);
    const float f_6 = pdf.get(6);
    const float f_7 = pdf.get(7);
    const float f_8 = pdf.get(8);
    const float f_9 = pdf.get(9);
    const float f_10 = pdf.get(10);
    const float f_11 = pdf.get(11);
    const float f_12 = pdf.get(12);
    const float f_13 = pdf.get(13);
    const float f_14 = pdf.get(14);
    const float f_15 = pdf.get(15);
    const float f_16 = pdf.get(16);
    const float f_17 = pdf.get(17);
    const float f_18 = pdf.get(18);
    const float vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const float vel1Term = f_1 + f_11 + f_15 + f_7;
    const float vel2Term = f_12 + f_13 + f_5;
    const float rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term +
                      vel1Term + vel2Term;
    const float u_0 = -force.get(0) * 0.50000000000000000f / rho + u[0];
    const float u_1 = -force.get(1) * 0.50000000000000000f / rho + u[1];
    const float u_2 = -force.get(2) * 0.50000000000000000f / rho + u[2];
    float u_new[3] = {u_0, u_1, u_2};

    Equilibrium::kernel_set_device(pdf, u_new, rho);
  }
}

void set(cuda::GPUField<float> *pdf_field, cuda::GPUField<float> *force_field,
         Vector3<float> const &u, Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<float> dev_data(u.data(), u.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*pdf_field, ci));
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*force_field, ci));
  kernel.addParam(const_cast<const float *>(dev_data_ptr));
  kernel();
}
} // namespace Velocity

namespace MomentumDensity {
__global__ void kernel_sum(cuda::FieldAccessor<float> pdf,
                           cuda::FieldAccessor<float> force,
                           float *RESTRICT const out) {
  pdf.set(blockIdx, threadIdx);
  force.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    const uint bufsize = 3u;
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, bufsize);
    const float f_0 = pdf.get(0);
    const float f_1 = pdf.get(1);
    const float f_2 = pdf.get(2);
    const float f_3 = pdf.get(3);
    const float f_4 = pdf.get(4);
    const float f_5 = pdf.get(5);
    const float f_6 = pdf.get(6);
    const float f_7 = pdf.get(7);
    const float f_8 = pdf.get(8);
    const float f_9 = pdf.get(9);
    const float f_10 = pdf.get(10);
    const float f_11 = pdf.get(11);
    const float f_12 = pdf.get(12);
    const float f_13 = pdf.get(13);
    const float f_14 = pdf.get(14);
    const float f_15 = pdf.get(15);
    const float f_16 = pdf.get(16);
    const float f_17 = pdf.get(17);
    const float f_18 = pdf.get(18);
    const float vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const float momdensity_0 = -f_13 - f_17 - f_3 - f_7 - f_9 + vel0Term;
    const float vel1Term = f_1 + f_11 + f_15 + f_7;
    const float momdensity_1 = -f_10 - f_12 - f_16 - f_2 + f_8 - f_9 + vel1Term;
    const float vel2Term = f_12 + f_13 + f_5;
    const float momdensity_2 =
        f_11 + f_14 - f_15 - f_16 - f_17 - f_18 - f_6 + vel2Term;
    const float md_0 = force.get(0) * 0.50000000000000000f + momdensity_0;
    const float md_1 = force.get(1) * 0.50000000000000000f + momdensity_1;
    const float md_2 = force.get(2) * 0.50000000000000000f + momdensity_2;
    out[bufsize * offset + 0u] += md_0;
    out[bufsize * offset + 1u] += md_1;
    out[bufsize * offset + 2u] += md_2;
  }
}

Vector3<float> reduce(cuda::GPUField<float> const *pdf_field,
                      cuda::GPUField<float> const *force_field) {
  thrust::device_vector<float> dev_data(3u, float{0});
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  WALBERLA_FOR_ALL_CELLS_XYZ(pdf_field, {
    Cell cell(x, y, z);
    CellInterval ci(cell, cell);
    auto kernel = cuda::make_kernel(kernel_sum);
    kernel.addFieldIndexingParam(
        cuda::FieldIndexing<float>::interval(*pdf_field, ci));
    kernel.addFieldIndexingParam(
        cuda::FieldIndexing<float>::interval(*force_field, ci));
    kernel.addParam(dev_data_ptr);
    kernel();
  });
  Vector3<float> mom(float{0});
  thrust::copy(dev_data.begin(), dev_data.begin() + 3u, mom.data());
  return mom;
}
} // namespace MomentumDensity

namespace PressureTensor {
__global__ void kernel_get(cuda::FieldAccessor<float> pdf,
                           float *RESTRICT const out) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    const uint bufsize = 9u;
    const uint offset =
        getLinearIndexFZYX(blockIdx, threadIdx, gridDim, blockDim, bufsize);
    const float f_0 = pdf.get(0);
    const float f_1 = pdf.get(1);
    const float f_2 = pdf.get(2);
    const float f_3 = pdf.get(3);
    const float f_4 = pdf.get(4);
    const float f_5 = pdf.get(5);
    const float f_6 = pdf.get(6);
    const float f_7 = pdf.get(7);
    const float f_8 = pdf.get(8);
    const float f_9 = pdf.get(9);
    const float f_10 = pdf.get(10);
    const float f_11 = pdf.get(11);
    const float f_12 = pdf.get(12);
    const float f_13 = pdf.get(13);
    const float f_14 = pdf.get(14);
    const float f_15 = pdf.get(15);
    const float f_16 = pdf.get(16);
    const float f_17 = pdf.get(17);
    const float f_18 = pdf.get(18);
    const float p_0 =
        f_10 + f_13 + f_14 + f_17 + f_18 + f_3 + f_4 + f_7 + f_8 + f_9;
    const float p_1 = -f_10 - f_7 + f_8 + f_9;
    const float p_2 = -f_13 + f_14 + f_17 - f_18;
    const float p_3 = -f_10 - f_7 + f_8 + f_9;
    const float p_4 =
        f_1 + f_10 + f_11 + f_12 + f_15 + f_16 + f_2 + f_7 + f_8 + f_9;
    const float p_5 = f_11 - f_12 - f_15 + f_16;
    const float p_6 = -f_13 + f_14 + f_17 - f_18;
    const float p_7 = f_11 - f_12 - f_15 + f_16;
    const float p_8 =
        f_11 + f_12 + f_13 + f_14 + f_15 + f_16 + f_17 + f_18 + f_5 + f_6;
    out[bufsize * offset + 0u] = p_0;
    out[bufsize * offset + 1u] = p_1;
    out[bufsize * offset + 2u] = p_2;

    out[bufsize * offset + 3u] = p_3;
    out[bufsize * offset + 4u] = p_4;
    out[bufsize * offset + 5u] = p_5;

    out[bufsize * offset + 6u] = p_6;
    out[bufsize * offset + 7u] = p_7;
    out[bufsize * offset + 8u] = p_8;
  }
}

Matrix3<float> get(cuda::GPUField<float> const *pdf_field, Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<float> dev_data(9u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = cuda::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(
      cuda::FieldIndexing<float>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  Matrix3<float> out;
  thrust::copy(dev_data.begin(), dev_data.begin() + 9u, out.data());
  return out;
}
} // namespace PressureTensor

} // namespace accessor
} // namespace lbm
} // namespace walberla
