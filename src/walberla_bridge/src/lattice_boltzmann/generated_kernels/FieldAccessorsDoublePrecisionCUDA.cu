#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023-2024 The ESPResSo project
 * Copyright (C) 2020 The waLBerla project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

// kernel generated with pystencils v1.3.3, lbmpy v1.3.3, lbmpy_walberla/pystencils_walberla from waLBerla commit b0842e1a493ce19ef1bbb8d2cf382fc343970a7f

/**
 * @file
 * Lattice field accessors.
 * Adapted from the waLBerla source file
 * https://i10git.cs.fau.de/walberla/walberla/-/blob/a16141524c58ab88386e2a0f8fdd7c63c5edd704/python/lbmpy_walberla/templates/LatticeModel.tmpl.h
 */

#include <core/DataTypes.h>
#include <core/cell/Cell.h>
#include <core/cell/CellInterval.h>
#include <core/math/Matrix3.h>
#include <core/math/Vector3.h>

#include <field/iterators/IteratorMacros.h>

#include <gpu/FieldAccessor.h>
#include <gpu/FieldIndexing.h>
#include <gpu/GPUField.h>
#include <gpu/Kernel.h>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

#include <array>
#include <vector>

#if defined(__NVCC__)
#define RESTRICT __restrict__
#if defined(__NVCC_DIAG_PRAGMA_SUPPORT__)
#pragma nv_diagnostic push
#pragma nv_diag_suppress 177 // unused variable
#else
#pragma push
#pragma diag_suppress 177 // unused variable
#endif
#elif defined(__clang__)
#if defined(__CUDA__)
#if defined(__CUDA_ARCH__)
// clang compiling CUDA code in device mode
#define RESTRICT __restrict__
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wunused-variable"
#pragma clang diagnostic ignored "-Wunused-parameter"
#else
// clang compiling CUDA code in host mode
#define RESTRICT __restrict__
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wunused-variable"
#pragma clang diagnostic ignored "-Wunused-parameter"
#endif
#endif
#elif defined(__GNUC__) or defined(__GNUG__)
#define RESTRICT __restrict__
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wunused-variable"
#pragma GCC diagnostic ignored "-Wunused-parameter"
#elif defined(_MSC_VER)
#define RESTRICT __restrict
#else
#define RESTRICT
#endif

/** @brief Get linear index of flattened data with original layout @c fzyx. */
static __forceinline__ __device__ uint getLinearIndex(uint3 blockIdx, uint3 threadIdx, uint3 gridDim, uint3 blockDim, uint fOffset) {
  auto const x = threadIdx.x;
  auto const y = blockIdx.x;
  auto const z = blockIdx.y;
  auto const f = blockIdx.z;
  auto const ySize = gridDim.x;
  auto const zSize = gridDim.y;
  auto const fSize = fOffset;
  return f +
         z * fSize +
         y * fSize * zSize +
         x * fSize * zSize * ySize;
}

namespace walberla {
namespace lbm {
namespace accessor {

namespace Population {
__global__ void kernel_get_interval(
    gpu::FieldAccessor<double> pdf,
    double *RESTRICT const pop) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    uint const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 19u);
    pop[offset + 0u] = pdf.get(0u);
    pop[offset + 1u] = pdf.get(1u);
    pop[offset + 2u] = pdf.get(2u);
    pop[offset + 3u] = pdf.get(3u);
    pop[offset + 4u] = pdf.get(4u);
    pop[offset + 5u] = pdf.get(5u);
    pop[offset + 6u] = pdf.get(6u);
    pop[offset + 7u] = pdf.get(7u);
    pop[offset + 8u] = pdf.get(8u);
    pop[offset + 9u] = pdf.get(9u);
    pop[offset + 10u] = pdf.get(10u);
    pop[offset + 11u] = pdf.get(11u);
    pop[offset + 12u] = pdf.get(12u);
    pop[offset + 13u] = pdf.get(13u);
    pop[offset + 14u] = pdf.get(14u);
    pop[offset + 15u] = pdf.get(15u);
    pop[offset + 16u] = pdf.get(16u);
    pop[offset + 17u] = pdf.get(17u);
    pop[offset + 18u] = pdf.get(18u);
  }
}

__global__ void kernel_get(
    gpu::FieldAccessor<double> pdf,
    double *RESTRICT const pop) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    pop[0u] = pdf.get(0u);
    pop[1u] = pdf.get(1u);
    pop[2u] = pdf.get(2u);
    pop[3u] = pdf.get(3u);
    pop[4u] = pdf.get(4u);
    pop[5u] = pdf.get(5u);
    pop[6u] = pdf.get(6u);
    pop[7u] = pdf.get(7u);
    pop[8u] = pdf.get(8u);
    pop[9u] = pdf.get(9u);
    pop[10u] = pdf.get(10u);
    pop[11u] = pdf.get(11u);
    pop[12u] = pdf.get(12u);
    pop[13u] = pdf.get(13u);
    pop[14u] = pdf.get(14u);
    pop[15u] = pdf.get(15u);
    pop[16u] = pdf.get(16u);
    pop[17u] = pdf.get(17u);
    pop[18u] = pdf.get(18u);
  }
}

__global__ void kernel_set_interval(
    gpu::FieldAccessor<double> pdf,
    double const *RESTRICT const pop) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    uint const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 19u);
    pdf.get(0u) = pop[offset + 0u];
    pdf.get(1u) = pop[offset + 1u];
    pdf.get(2u) = pop[offset + 2u];
    pdf.get(3u) = pop[offset + 3u];
    pdf.get(4u) = pop[offset + 4u];
    pdf.get(5u) = pop[offset + 5u];
    pdf.get(6u) = pop[offset + 6u];
    pdf.get(7u) = pop[offset + 7u];
    pdf.get(8u) = pop[offset + 8u];
    pdf.get(9u) = pop[offset + 9u];
    pdf.get(10u) = pop[offset + 10u];
    pdf.get(11u) = pop[offset + 11u];
    pdf.get(12u) = pop[offset + 12u];
    pdf.get(13u) = pop[offset + 13u];
    pdf.get(14u) = pop[offset + 14u];
    pdf.get(15u) = pop[offset + 15u];
    pdf.get(16u) = pop[offset + 16u];
    pdf.get(17u) = pop[offset + 17u];
    pdf.get(18u) = pop[offset + 18u];
  }
}

__global__ void kernel_set(
    gpu::FieldAccessor<double> pdf,
    double const *RESTRICT const pop) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    pdf.get(0u) = pop[0u];
    pdf.get(1u) = pop[1u];
    pdf.get(2u) = pop[2u];
    pdf.get(3u) = pop[3u];
    pdf.get(4u) = pop[4u];
    pdf.get(5u) = pop[5u];
    pdf.get(6u) = pop[6u];
    pdf.get(7u) = pop[7u];
    pdf.get(8u) = pop[8u];
    pdf.get(9u) = pop[9u];
    pdf.get(10u) = pop[10u];
    pdf.get(11u) = pop[11u];
    pdf.get(12u) = pop[12u];
    pdf.get(13u) = pop[13u];
    pdf.get(14u) = pop[14u];
    pdf.get(15u) = pop[15u];
    pdf.get(16u) = pop[16u];
    pdf.get(17u) = pop[17u];
    pdf.get(18u) = pop[18u];
  }
}

std::array<double, 19u> get(
    gpu::GPUField<double> const *pdf_field,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(19u, double{0});
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::array<double, 19u> pop;
  thrust::copy(dev_data.begin(), dev_data.end(), pop.data());
  return pop;
}

void set(
    gpu::GPUField<double> *pdf_field,
    std::array<double, 19u> const &pop,
    Cell const &cell) {
  thrust::device_vector<double> dev_data(pop.data(), pop.data() + 19u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  CellInterval ci(cell, cell);
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

void initialize(
    gpu::GPUField<double> *pdf_field,
    std::array<double, 19u> const &pop) {
  CellInterval ci = pdf_field->xyzSizeWithGhostLayer();
  thrust::device_vector<double> dev_data(pop.data(), pop.data() + 19u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

std::vector<double> get(
    gpu::GPUField<double> const *pdf_field,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(ci.numCells() * 19u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get_interval);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::vector<double> out(ci.numCells() * 19u);
  thrust::copy(dev_data.begin(), dev_data.end(), out.data());
  return out;
}

void set(
    gpu::GPUField<double> *pdf_field,
    std::vector<double> const &values,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(values.begin(), values.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set_interval);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}
} // namespace Population

namespace Vector {
__global__ void kernel_get_interval(
    gpu::FieldAccessor<double> vec,
    double *const out) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    uint const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 3u);
    out[offset + 0u] = vec.get(0u);
    out[offset + 1u] = vec.get(1u);
    out[offset + 2u] = vec.get(2u);
  }
}

__global__ void kernel_get(
    gpu::FieldAccessor<double> vec,
    double *const out) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    out[0u] = vec.get(0u);
    out[1u] = vec.get(1u);
    out[2u] = vec.get(2u);
  }
}

__global__ void kernel_set_interval(
    gpu::FieldAccessor<double> vec,
    double const *RESTRICT const u) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    uint const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 3u);
    vec.get(0u) = u[offset + 0u];
    vec.get(1u) = u[offset + 1u];
    vec.get(2u) = u[offset + 2u];
  }
}

__global__ void kernel_set(
    gpu::FieldAccessor<double> vec,
    const double *RESTRICT const u) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    vec.get(0u) = u[0u];
    vec.get(1u) = u[1u];
    vec.get(2u) = u[2u];
  }
}

__global__ void kernel_add_interval(
    gpu::FieldAccessor<double> vec,
    double const *RESTRICT const u) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    uint const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, 3u);
    vec.get(0u) += u[offset + 0u];
    vec.get(1u) += u[offset + 1u];
    vec.get(2u) += u[offset + 2u];
  }
}

__global__ void kernel_add(
    gpu::FieldAccessor<double> vec,
    double const *RESTRICT const u) {
  vec.set(blockIdx, threadIdx);
  if (vec.isValidPosition()) {
    vec.get(0u) += u[0u];
    vec.get(1u) += u[1u];
    vec.get(2u) += u[2u];
  }
}

Vector3<double> get(
    gpu::GPUField<double> const *vec_field,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  Vector3<double> vec;
  thrust::copy(dev_data.begin(), dev_data.end(), vec.data());
  return vec;
}

void set(
    gpu::GPUField<double> *vec_field,
    Vector3<double> const &vec,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(vec.data(), vec.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

void add(
    gpu::GPUField<double> *vec_field,
    Vector3<double> const &vec,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(vec.data(), vec.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_add);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

void initialize(
    gpu::GPUField<double> *vec_field,
    Vector3<double> const &vec) {
  CellInterval ci = vec_field->xyzSizeWithGhostLayer();
  thrust::device_vector<double> dev_data(vec.data(), vec.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

void add_to_all(
    gpu::GPUField<double> *vec_field,
    Vector3<double> const &vec) {
  CellInterval ci = vec_field->xyzSizeWithGhostLayer();
  thrust::device_vector<double> dev_data(vec.data(), vec.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_add);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

std::vector<double> get(
    gpu::GPUField<double> const *vec_field,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(ci.numCells() * 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get_interval);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::vector<double> out(ci.numCells() * 3u);
  thrust::copy(dev_data.begin(), dev_data.end(), out.data());
  return out;
}

void set(
    gpu::GPUField<double> *vec_field,
    std::vector<double> const &values,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(values.begin(), values.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set_interval);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*vec_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}
} // namespace Vector

namespace Interpolation {
/** @brief Calculate interpolation weights. */
static __forceinline__ __device__ void calculate_weights(
    double const *RESTRICT const pos,
    int *RESTRICT const corner,
    double *RESTRICT const weights,
    uint gl) {
#pragma unroll
  for (int dim = 0; dim < 3; ++dim) {
    auto const fractional_index = pos[dim] - double{0.5};
    auto const nmp = floorf(fractional_index);
    auto const distance = fractional_index - nmp - double{0.5};
    corner[dim] = __double2int_rn(nmp) + static_cast<int>(gl);
    weights[dim * 2 + 0] = double{0.5} - distance;
    weights[dim * 2 + 1] = double{0.5} + distance;
  }
}

__global__ void kernel_get(
    gpu::FieldAccessor<double> vec,
    double const *RESTRICT const pos,
    double *RESTRICT const vel,
    uint n_pos,
    uint gl) {

  uint pos_index = blockIdx.y * gridDim.x * blockDim.x +
                   blockDim.x * blockIdx.x + threadIdx.x;

  vec.set({0u, 0u, 0u}, {0u, 0u, 0u});
  if (vec.isValidPosition() and pos_index < n_pos) {
    auto const array_offset = pos_index * uint(3u);
    int corner[3];
    double weights[3][2];
    calculate_weights(pos + array_offset, corner, &weights[0][0], gl);
#pragma unroll
    for (int i = 0; i < 2; i++) {
      auto const cx = corner[0] + i;
      auto const wx = weights[0][i];
#pragma unroll
      for (int j = 0; j < 2; j++) {
        auto const cy = corner[1] + j;
        auto const wxy = wx * weights[1][j];
#pragma unroll
        for (int k = 0; k < 2; k++) {
          auto const cz = corner[2] + k;
          auto const weight = wxy * weights[2][k];
          vel[array_offset + 0u] += weight * vec.getNeighbor(cx, cy, cz, 0u);
          vel[array_offset + 1u] += weight * vec.getNeighbor(cx, cy, cz, 1u);
          vel[array_offset + 2u] += weight * vec.getNeighbor(cx, cy, cz, 2u);
        }
      }
    }
  }
}

__global__ void kernel_set(
    gpu::FieldAccessor<double> vec,
    double const *RESTRICT const pos,
    double const *RESTRICT const forces,
    uint n_pos,
    uint gl) {

  uint pos_index = blockIdx.y * gridDim.x * blockDim.x +
                   blockDim.x * blockIdx.x + threadIdx.x;

  vec.set({0u, 0u, 0u}, {0u, 0u, 0u});
  if (vec.isValidPosition() and pos_index < n_pos) {
    auto const array_offset = pos_index * uint(3u);
    int corner[3];
    double weights[3][2];
    calculate_weights(pos + array_offset, corner, &weights[0][0], gl);
#pragma unroll
    for (int i = 0; i < 2; i++) {
      auto const cx = corner[0] + i;
      auto const wx = weights[0][i];
#pragma unroll
      for (int j = 0; j < 2; j++) {
        auto const cy = corner[1] + j;
        auto const wxy = wx * weights[1][j];
#pragma unroll
        for (int k = 0; k < 2; k++) {
          auto const cz = corner[2] + k;
          auto const weight = wxy * weights[2][k];
          atomicAdd(&vec.getNeighbor(cx, cy, cz, 0u),
                    weight * forces[array_offset + 0u]);
          atomicAdd(&vec.getNeighbor(cx, cy, cz, 1u),
                    weight * forces[array_offset + 1u]);
          atomicAdd(&vec.getNeighbor(cx, cy, cz, 2u),
                    weight * forces[array_offset + 2u]);
        }
      }
    }
  }
}

static dim3 calculate_dim_grid(uint const threads_x,
                               uint const blocks_per_grid_y,
                               uint const threads_per_block) {
  assert(threads_x >= 1u);
  assert(blocks_per_grid_y >= 1u);
  assert(threads_per_block >= 1u);
  auto const threads_y = threads_per_block * blocks_per_grid_y;
  auto const blocks_per_grid_x = (threads_x + threads_y - 1) / threads_y;
  return make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);
}

std::vector<double>
get(
    gpu::GPUField<double> const *vec_field,
    std::vector<double> const &pos,
    uint gl) {
  thrust::device_vector<double> dev_pos(pos.begin(), pos.end());
  thrust::device_vector<double> dev_vel(pos.size());
  auto const dev_pos_ptr = thrust::raw_pointer_cast(dev_pos.data());
  auto const dev_vel_ptr = thrust::raw_pointer_cast(dev_vel.data());

  auto const threads_per_block = uint(64u);
  auto const n_pos = static_cast<uint>(pos.size() / 3ul);
  auto const dim_grid = calculate_dim_grid(n_pos, 4u, threads_per_block);
  kernel_get<<<dim_grid, threads_per_block, 0u, nullptr>>>(
      gpu::FieldIndexing<double>::withGhostLayerXYZ(*vec_field, gl).gpuAccess(),
      dev_pos_ptr, dev_vel_ptr, n_pos, gl);

  std::vector<double> out(pos.size());
  thrust::copy(dev_vel.begin(), dev_vel.end(), out.data());
  return out;
}

void set(
    gpu::GPUField<double> const *vec_field,
    std::vector<double> const &pos,
    std::vector<double> const &forces,
    uint gl) {
  thrust::device_vector<double> dev_pos(pos.begin(), pos.end());
  thrust::device_vector<double> dev_for(forces.begin(), forces.end());
  auto const dev_pos_ptr = thrust::raw_pointer_cast(dev_pos.data());
  auto const dev_for_ptr = thrust::raw_pointer_cast(dev_for.data());

  auto const threads_per_block = uint(64u);
  auto const n_pos = static_cast<uint>(pos.size() / 3ul);
  auto const dim_grid = calculate_dim_grid(n_pos, 4u, threads_per_block);
  kernel_set<<<dim_grid, threads_per_block, 0u, nullptr>>>(
      gpu::FieldIndexing<double>::withGhostLayerXYZ(*vec_field, gl).gpuAccess(),
      dev_pos_ptr, dev_for_ptr, n_pos, gl);
}
} // namespace Interpolation

namespace Equilibrium {
__device__ void kernel_set_device(
    gpu::FieldAccessor<double> pdf,
    double const *RESTRICT const u,
    double rho) {

  pdf.get(0u) = rho * -0.33333333333333331 * (u[0] * u[0]) + rho * -0.33333333333333331 * (u[1] * u[1]) + rho * -0.33333333333333331 * (u[2] * u[2]) + rho * 0.33333333333333331;
  pdf.get(1u) = rho * -0.16666666666666666 * (u[0] * u[0]) + rho * -0.16666666666666666 * (u[2] * u[2]) + rho * 0.055555555555555552 + rho * 0.16666666666666666 * u[1] + rho * 0.16666666666666666 * (u[1] * u[1]);
  pdf.get(2u) = rho * -0.16666666666666666 * u[1] + rho * -0.16666666666666666 * (u[0] * u[0]) + rho * -0.16666666666666666 * (u[2] * u[2]) + rho * 0.055555555555555552 + rho * 0.16666666666666666 * (u[1] * u[1]);
  pdf.get(3u) = rho * -0.16666666666666666 * u[0] + rho * -0.16666666666666666 * (u[1] * u[1]) + rho * -0.16666666666666666 * (u[2] * u[2]) + rho * 0.055555555555555552 + rho * 0.16666666666666666 * (u[0] * u[0]);
  pdf.get(4u) = rho * -0.16666666666666666 * (u[1] * u[1]) + rho * -0.16666666666666666 * (u[2] * u[2]) + rho * 0.055555555555555552 + rho * 0.16666666666666666 * u[0] + rho * 0.16666666666666666 * (u[0] * u[0]);
  pdf.get(5u) = rho * -0.16666666666666666 * (u[0] * u[0]) + rho * -0.16666666666666666 * (u[1] * u[1]) + rho * 0.055555555555555552 + rho * 0.16666666666666666 * u[2] + rho * 0.16666666666666666 * (u[2] * u[2]);
  pdf.get(6u) = rho * -0.16666666666666666 * u[2] + rho * -0.16666666666666666 * (u[0] * u[0]) + rho * -0.16666666666666666 * (u[1] * u[1]) + rho * 0.055555555555555552 + rho * 0.16666666666666666 * (u[2] * u[2]);
  pdf.get(7u) = rho * -0.083333333333333329 * u[0] + rho * -0.25 * u[0] * u[1] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[1] + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[1] * u[1]);
  pdf.get(8u) = rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[0] + rho * 0.083333333333333329 * u[1] + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[1] * u[1]) + rho * 0.25 * u[0] * u[1];
  pdf.get(9u) = rho * -0.083333333333333329 * u[0] + rho * -0.083333333333333329 * u[1] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[1] * u[1]) + rho * 0.25 * u[0] * u[1];
  pdf.get(10u) = rho * -0.083333333333333329 * u[1] + rho * -0.25 * u[0] * u[1] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[0] + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[1] * u[1]);
  pdf.get(11u) = rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[1] + rho * 0.083333333333333329 * u[2] + rho * 0.083333333333333329 * (u[1] * u[1]) + rho * 0.083333333333333329 * (u[2] * u[2]) + rho * 0.25 * u[1] * u[2];
  pdf.get(12u) = rho * -0.083333333333333329 * u[1] + rho * -0.25 * u[1] * u[2] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[2] + rho * 0.083333333333333329 * (u[1] * u[1]) + rho * 0.083333333333333329 * (u[2] * u[2]);
  pdf.get(13u) = rho * -0.083333333333333329 * u[0] + rho * -0.25 * u[0] * u[2] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[2] + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[2] * u[2]);
  pdf.get(14u) = rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[0] + rho * 0.083333333333333329 * u[2] + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[2] * u[2]) + rho * 0.25 * u[0] * u[2];
  pdf.get(15u) = rho * -0.083333333333333329 * u[2] + rho * -0.25 * u[1] * u[2] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[1] + rho * 0.083333333333333329 * (u[1] * u[1]) + rho * 0.083333333333333329 * (u[2] * u[2]);
  pdf.get(16u) = rho * -0.083333333333333329 * u[1] + rho * -0.083333333333333329 * u[2] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * (u[1] * u[1]) + rho * 0.083333333333333329 * (u[2] * u[2]) + rho * 0.25 * u[1] * u[2];
  pdf.get(17u) = rho * -0.083333333333333329 * u[0] + rho * -0.083333333333333329 * u[2] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[2] * u[2]) + rho * 0.25 * u[0] * u[2];
  pdf.get(18u) = rho * -0.083333333333333329 * u[2] + rho * -0.25 * u[0] * u[2] + rho * 0.027777777777777776 + rho * 0.083333333333333329 * u[0] + rho * 0.083333333333333329 * (u[0] * u[0]) + rho * 0.083333333333333329 * (u[2] * u[2]);
}
} // namespace Equilibrium

namespace Density {
__global__ void kernel_get(
    gpu::FieldAccessor<double> pdf,
    double *RESTRICT const out) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    uint const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, uint(1u));
    double const f_0 = pdf.get(0u);
    double const f_1 = pdf.get(1u);
    double const f_2 = pdf.get(2u);
    double const f_3 = pdf.get(3u);
    double const f_4 = pdf.get(4u);
    double const f_5 = pdf.get(5u);
    double const f_6 = pdf.get(6u);
    double const f_7 = pdf.get(7u);
    double const f_8 = pdf.get(8u);
    double const f_9 = pdf.get(9u);
    double const f_10 = pdf.get(10u);
    double const f_11 = pdf.get(11u);
    double const f_12 = pdf.get(12u);
    double const f_13 = pdf.get(13u);
    double const f_14 = pdf.get(14u);
    double const f_15 = pdf.get(15u);
    double const f_16 = pdf.get(16u);
    double const f_17 = pdf.get(17u);
    double const f_18 = pdf.get(18u);
    const double vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const double vel1Term = f_1 + f_11 + f_15 + f_7;
    const double vel2Term = f_12 + f_13 + f_5;
    const double rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term + vel1Term + vel2Term;
    out[offset] = rho;
  }
}

__global__ void kernel_set(
    gpu::FieldAccessor<double> pdf,
    double const *RESTRICT const rho_in) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    uint const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, uint(1u));
    double const f_0 = pdf.get(0u);
    double const f_1 = pdf.get(1u);
    double const f_2 = pdf.get(2u);
    double const f_3 = pdf.get(3u);
    double const f_4 = pdf.get(4u);
    double const f_5 = pdf.get(5u);
    double const f_6 = pdf.get(6u);
    double const f_7 = pdf.get(7u);
    double const f_8 = pdf.get(8u);
    double const f_9 = pdf.get(9u);
    double const f_10 = pdf.get(10u);
    double const f_11 = pdf.get(11u);
    double const f_12 = pdf.get(12u);
    double const f_13 = pdf.get(13u);
    double const f_14 = pdf.get(14u);
    double const f_15 = pdf.get(15u);
    double const f_16 = pdf.get(16u);
    double const f_17 = pdf.get(17u);
    double const f_18 = pdf.get(18u);
    const double vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const double momdensity_0 = -f_13 - f_17 - f_3 - f_7 - f_9 + vel0Term;
    const double vel1Term = f_1 + f_11 + f_15 + f_7;
    const double momdensity_1 = -f_10 - f_12 - f_16 - f_2 + f_8 - f_9 + vel1Term;
    const double vel2Term = f_12 + f_13 + f_5;
    const double momdensity_2 = f_11 + f_14 - f_15 - f_16 - f_17 - f_18 - f_6 + vel2Term;
    const double rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term + vel1Term + vel2Term;

    // calculate current velocity (before density change)
    double const conversion = double(1) / rho;
    double const u_old[3] = {momdensity_0 * conversion, momdensity_1 * conversion, momdensity_2 * conversion};

    Equilibrium::kernel_set_device(pdf, u_old, rho_in[offset]);
  }
}

double get(
    gpu::GPUField<double> const *pdf_field,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(1u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  double rho = dev_data[0u];
  return rho;
}

void set(
    gpu::GPUField<double> *pdf_field,
    const double rho,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(1u, rho);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}

std::vector<double> get(
    gpu::GPUField<double> const *pdf_field,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(ci.numCells());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  std::vector<double> out(ci.numCells());
  thrust::copy(dev_data.begin(), dev_data.end(), out.begin());
  return out;
}

void set(
    gpu::GPUField<double> *pdf_field,
    std::vector<double> const &values,
    CellInterval const &ci) {
  thrust::device_vector<double> dev_data(values.begin(), values.end());
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}
} // namespace Density

namespace Velocity {
__global__ void kernel_set(
    gpu::FieldAccessor<double> pdf,
    gpu::FieldAccessor<double> force,
    double const *RESTRICT const u_in) {
  pdf.set(blockIdx, threadIdx);
  force.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    uint const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, uint(3u));
    uint const bufsize = 3u;
    double const *RESTRICT const u = u_in + bufsize * offset;
    double const f_0 = pdf.get(0u);
    double const f_1 = pdf.get(1u);
    double const f_2 = pdf.get(2u);
    double const f_3 = pdf.get(3u);
    double const f_4 = pdf.get(4u);
    double const f_5 = pdf.get(5u);
    double const f_6 = pdf.get(6u);
    double const f_7 = pdf.get(7u);
    double const f_8 = pdf.get(8u);
    double const f_9 = pdf.get(9u);
    double const f_10 = pdf.get(10u);
    double const f_11 = pdf.get(11u);
    double const f_12 = pdf.get(12u);
    double const f_13 = pdf.get(13u);
    double const f_14 = pdf.get(14u);
    double const f_15 = pdf.get(15u);
    double const f_16 = pdf.get(16u);
    double const f_17 = pdf.get(17u);
    double const f_18 = pdf.get(18u);
    const double vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const double vel1Term = f_1 + f_11 + f_15 + f_7;
    const double vel2Term = f_12 + f_13 + f_5;
    const double rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term + vel1Term + vel2Term;
    const double u_0 = -force.get(0) * 0.50000000000000000 / rho + u[0];
    const double u_1 = -force.get(1) * 0.50000000000000000 / rho + u[1];
    const double u_2 = -force.get(2) * 0.50000000000000000 / rho + u[2];
    double u_new[3] = {u_0, u_1, u_2};

    Equilibrium::kernel_set_device(pdf, u_new, rho);
  }
}

void set(
    gpu::GPUField<double> *pdf_field,
    gpu::GPUField<double> *force_field,
    Vector3<double> const &u,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(u.data(), u.data() + 3u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_set);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*force_field, ci));
  kernel.addParam(const_cast<const double *>(dev_data_ptr));
  kernel();
}
} // namespace Velocity

namespace MomentumDensity {
__global__ void kernel_sum(
    gpu::FieldAccessor<double> pdf,
    gpu::FieldAccessor<double> force,
    double *RESTRICT const out) {
  pdf.set(blockIdx, threadIdx);
  force.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    uint const bufsize = 3u;
    uint const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, bufsize);
    double const f_0 = pdf.get(0u);
    double const f_1 = pdf.get(1u);
    double const f_2 = pdf.get(2u);
    double const f_3 = pdf.get(3u);
    double const f_4 = pdf.get(4u);
    double const f_5 = pdf.get(5u);
    double const f_6 = pdf.get(6u);
    double const f_7 = pdf.get(7u);
    double const f_8 = pdf.get(8u);
    double const f_9 = pdf.get(9u);
    double const f_10 = pdf.get(10u);
    double const f_11 = pdf.get(11u);
    double const f_12 = pdf.get(12u);
    double const f_13 = pdf.get(13u);
    double const f_14 = pdf.get(14u);
    double const f_15 = pdf.get(15u);
    double const f_16 = pdf.get(16u);
    double const f_17 = pdf.get(17u);
    double const f_18 = pdf.get(18u);
    const double vel0Term = f_10 + f_14 + f_18 + f_4 + f_8;
    const double momdensity_0 = -f_13 - f_17 - f_3 - f_7 - f_9 + vel0Term;
    const double vel1Term = f_1 + f_11 + f_15 + f_7;
    const double momdensity_1 = -f_10 - f_12 - f_16 - f_2 + f_8 - f_9 + vel1Term;
    const double vel2Term = f_12 + f_13 + f_5;
    const double momdensity_2 = f_11 + f_14 - f_15 - f_16 - f_17 - f_18 - f_6 + vel2Term;
    const double rho = f_0 + f_16 + f_17 + f_2 + f_3 + f_6 + f_9 + vel0Term + vel1Term + vel2Term;
    const double md_0 = force.get(0) * 0.50000000000000000 + momdensity_0;
    const double md_1 = force.get(1) * 0.50000000000000000 + momdensity_1;
    const double md_2 = force.get(2) * 0.50000000000000000 + momdensity_2;
    out[bufsize * offset + 0u] += md_0;
    out[bufsize * offset + 1u] += md_1;
    out[bufsize * offset + 2u] += md_2;
  }
}

Vector3<double> reduce(
    gpu::GPUField<double> const *pdf_field,
    gpu::GPUField<double> const *force_field) {
  thrust::device_vector<double> dev_data(3u, double{0});
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  WALBERLA_FOR_ALL_CELLS_XYZ(pdf_field, {
    Cell cell(x, y, z);
    CellInterval ci(cell, cell);
    auto kernel = gpu::make_kernel(kernel_sum);
    kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
    kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*force_field, ci));
    kernel.addParam(dev_data_ptr);
    kernel();
  });
  Vector3<double> mom(double{0});
  thrust::copy(dev_data.begin(), dev_data.begin() + 3u, mom.data());
  return mom;
}
} // namespace MomentumDensity

namespace PressureTensor {
__global__ void kernel_get(
    gpu::FieldAccessor<double> pdf,
    double *RESTRICT const out) {
  pdf.set(blockIdx, threadIdx);
  if (pdf.isValidPosition()) {
    uint const bufsize = 9u;
    uint const offset = getLinearIndex(blockIdx, threadIdx, gridDim, blockDim, bufsize);
    double const f_0 = pdf.get(0u);
    double const f_1 = pdf.get(1u);
    double const f_2 = pdf.get(2u);
    double const f_3 = pdf.get(3u);
    double const f_4 = pdf.get(4u);
    double const f_5 = pdf.get(5u);
    double const f_6 = pdf.get(6u);
    double const f_7 = pdf.get(7u);
    double const f_8 = pdf.get(8u);
    double const f_9 = pdf.get(9u);
    double const f_10 = pdf.get(10u);
    double const f_11 = pdf.get(11u);
    double const f_12 = pdf.get(12u);
    double const f_13 = pdf.get(13u);
    double const f_14 = pdf.get(14u);
    double const f_15 = pdf.get(15u);
    double const f_16 = pdf.get(16u);
    double const f_17 = pdf.get(17u);
    double const f_18 = pdf.get(18u);
    const double p_0 = f_10 + f_13 + f_14 + f_17 + f_18 + f_3 + f_4 + f_7 + f_8 + f_9;
    const double p_1 = -f_10 - f_7 + f_8 + f_9;
    const double p_2 = -f_13 + f_14 + f_17 - f_18;
    const double p_3 = -f_10 - f_7 + f_8 + f_9;
    const double p_4 = f_1 + f_10 + f_11 + f_12 + f_15 + f_16 + f_2 + f_7 + f_8 + f_9;
    const double p_5 = f_11 - f_12 - f_15 + f_16;
    const double p_6 = -f_13 + f_14 + f_17 - f_18;
    const double p_7 = f_11 - f_12 - f_15 + f_16;
    const double p_8 = f_11 + f_12 + f_13 + f_14 + f_15 + f_16 + f_17 + f_18 + f_5 + f_6;
    out[bufsize * offset + 0u] = p_0;
    out[bufsize * offset + 1u] = p_1;
    out[bufsize * offset + 2u] = p_2;

    out[bufsize * offset + 3u] = p_3;
    out[bufsize * offset + 4u] = p_4;
    out[bufsize * offset + 5u] = p_5;

    out[bufsize * offset + 6u] = p_6;
    out[bufsize * offset + 7u] = p_7;
    out[bufsize * offset + 8u] = p_8;
  }
}

Matrix3<double> get(
    gpu::GPUField<double> const *pdf_field,
    Cell const &cell) {
  CellInterval ci(cell, cell);
  thrust::device_vector<double> dev_data(9u);
  auto const dev_data_ptr = thrust::raw_pointer_cast(dev_data.data());
  auto kernel = gpu::make_kernel(kernel_get);
  kernel.addFieldIndexingParam(gpu::FieldIndexing<double>::interval(*pdf_field, ci));
  kernel.addParam(dev_data_ptr);
  kernel();
  Matrix3<double> out;
  thrust::copy(dev_data.begin(), dev_data.begin() + 9u, out.data());
  return out;
}
} // namespace PressureTensor

} // namespace accessor
} // namespace lbm
} // namespace walberla
