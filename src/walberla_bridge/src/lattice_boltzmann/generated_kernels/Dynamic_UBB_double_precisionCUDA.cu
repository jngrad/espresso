#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \\file Dynamic_UBB_double_precisionCUDA.cpp
//! \\author pystencils
//======================================================================================================================

// kernel generated with pystencils v1.3.3, lbmpy v1.3.3, lbmpy_walberla/pystencils_walberla from waLBerla commit b0842e1a493ce19ef1bbb8d2cf382fc343970a7f

#include "Dynamic_UBB_double_precisionCUDA.h"
#include "core/DataTypes.h"
#include "core/Macros.h"
#include "gpu/ErrorChecking.h"

#define FUNC_PREFIX __global__

using namespace std;

namespace walberla {
namespace lbm {

#if defined(__NVCC__)
#define RESTRICT __restrict__
#if defined(__NVCC_DIAG_PRAGMA_SUPPORT__)
#pragma nv_diagnostic push
#pragma nv_diag_suppress 177 // unused variable
#else
#pragma push
#pragma diag_suppress 177 // unused variable
#endif
#elif defined(__clang__)
#if defined(__CUDA__)
#if defined(__CUDA_ARCH__)
// clang compiling CUDA code in device mode
#define RESTRICT __restrict__
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wstrict-aliasing"
#pragma clang diagnostic ignored "-Wunused-variable"
#pragma clang diagnostic ignored "-Wconversion"
#pragma clang diagnostic ignored "-Wsign-compare"
#else
// clang compiling CUDA code in host mode
#define RESTRICT __restrict__
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wstrict-aliasing"
#pragma clang diagnostic ignored "-Wunused-variable"
#pragma clang diagnostic ignored "-Wconversion"
#pragma clang diagnostic ignored "-Wsign-compare"
#endif
#endif
#elif defined(__GNUC__) or defined(__GNUG__)
#define RESTRICT __restrict__
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wstrict-aliasing"
#pragma GCC diagnostic ignored "-Wunused-variable"
#pragma GCC diagnostic ignored "-Wconversion"
#elif defined(_MSC_VER)
#define RESTRICT __restrict
#else
#define RESTRICT
#endif

// NOLINTBEGIN(readability-non-const-parameter*)
namespace internal_dynamic_ubb_double_precisioncuda_boundary_Dynamic_UBB_double_precisionCUDA {
static FUNC_PREFIX __launch_bounds__(256) void dynamic_ubb_double_precisioncuda_boundary_Dynamic_UBB_double_precisionCUDA(uint8_t *RESTRICT const _data_indexVector, double *RESTRICT _data_pdfs, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3, int32_t indexVectorSize) {

  const int32_t f_in_inv_dir_idx[] = {0, 2, 1, 4, 3, 6, 5, 10, 9, 8, 7, 16, 15, 18, 17, 12, 11, 14, 13};

  const double weights[] = {0.33333333333333333, 0.055555555555555556, 0.055555555555555556, 0.055555555555555556, 0.055555555555555556, 0.055555555555555556, 0.055555555555555556, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778, 0.027777777777777778};

  const int32_t neighbour_offset_x[] = {0, 0, 0, -1, 1, 0, 0, -1, 1, -1, 1, 0, 0, -1, 1, 0, 0, -1, 1};
  const int32_t neighbour_offset_y[] = {0, 1, -1, 0, 0, 0, 0, 1, 1, -1, -1, 1, -1, 0, 0, 1, -1, 0, 0};
  const int32_t neighbour_offset_z[] = {0, 0, 0, 0, 0, 1, -1, 0, 0, 0, 0, 1, 1, 1, 1, -1, -1, -1, -1};

  if (blockDim.x * blockIdx.x + threadIdx.x < indexVectorSize) {
    uint8_t *RESTRICT _data_indexVector_10 = _data_indexVector;
    const int32_t x = *((int32_t *)(&_data_indexVector_10[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x]));
    uint8_t *RESTRICT _data_indexVector_14 = _data_indexVector + 4;
    const int32_t y = *((int32_t *)(&_data_indexVector_14[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x]));
    uint8_t *RESTRICT _data_indexVector_18 = _data_indexVector + 8;
    const int32_t z = *((int32_t *)(&_data_indexVector_18[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x]));
    uint8_t *RESTRICT _data_indexVector_112 = _data_indexVector + 12;
    const int32_t dir = *((int32_t *)(&_data_indexVector_112[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x]));
    double *RESTRICT _data_pdfs_10_2m1_318 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z - _stride_pdfs_2 + 18 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_34 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 4 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_11_20_38 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_1 + _stride_pdfs_2 * z + 8 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_1m1_20_310 = _data_pdfs + _stride_pdfs_1 * y - _stride_pdfs_1 + _stride_pdfs_2 * z + 10 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_21_314 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + _stride_pdfs_2 + 14 * _stride_pdfs_3;
    const double vel0Term = _data_pdfs_10_20_34[_stride_pdfs_0 * x + _stride_pdfs_0] + _data_pdfs_10_21_314[_stride_pdfs_0 * x + _stride_pdfs_0] + _data_pdfs_10_2m1_318[_stride_pdfs_0 * x + _stride_pdfs_0] + _data_pdfs_11_20_38[_stride_pdfs_0 * x + _stride_pdfs_0] + _data_pdfs_1m1_20_310[_stride_pdfs_0 * x + _stride_pdfs_0];
    double *RESTRICT _data_pdfs_11_2m1_315 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_1 + _stride_pdfs_2 * z - _stride_pdfs_2 + 15 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_11_20_37 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_1 + _stride_pdfs_2 * z + 7 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_11_20_31 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_1 + _stride_pdfs_2 * z + _stride_pdfs_3;
    double *RESTRICT _data_pdfs_11_21_311 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_1 + _stride_pdfs_2 * z + _stride_pdfs_2 + 11 * _stride_pdfs_3;
    const double vel1Term = _data_pdfs_11_20_31[_stride_pdfs_0 * x] + _data_pdfs_11_20_37[_stride_pdfs_0 * x - _stride_pdfs_0] + _data_pdfs_11_21_311[_stride_pdfs_0 * x] + _data_pdfs_11_2m1_315[_stride_pdfs_0 * x];
    double *RESTRICT _data_pdfs_1m1_21_312 = _data_pdfs + _stride_pdfs_1 * y - _stride_pdfs_1 + _stride_pdfs_2 * z + _stride_pdfs_2 + 12 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_21_313 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + _stride_pdfs_2 + 13 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_21_35 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + _stride_pdfs_2 + 5 * _stride_pdfs_3;
    const double vel2Term = _data_pdfs_10_21_313[_stride_pdfs_0 * x - _stride_pdfs_0] + _data_pdfs_10_21_35[_stride_pdfs_0 * x] + _data_pdfs_1m1_21_312[_stride_pdfs_0 * x];
    double *RESTRICT _data_pdfs_1m1_2m1_316 = _data_pdfs + _stride_pdfs_1 * y - _stride_pdfs_1 + _stride_pdfs_2 * z - _stride_pdfs_2 + 16 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_2m1_317 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z - _stride_pdfs_2 + 17 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_2m1_36 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z - _stride_pdfs_2 + 6 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_30 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z;
    double *RESTRICT _data_pdfs_1m1_20_39 = _data_pdfs + _stride_pdfs_1 * y - _stride_pdfs_1 + _stride_pdfs_2 * z + 9 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_1m1_20_32 = _data_pdfs + _stride_pdfs_1 * y - _stride_pdfs_1 + _stride_pdfs_2 * z + 2 * _stride_pdfs_3;
    double *RESTRICT _data_pdfs_10_20_33 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 3 * _stride_pdfs_3;
    const double rho = vel0Term + vel1Term + vel2Term + _data_pdfs_10_20_30[_stride_pdfs_0 * x] + _data_pdfs_10_20_33[_stride_pdfs_0 * x - _stride_pdfs_0] + _data_pdfs_10_2m1_317[_stride_pdfs_0 * x - _stride_pdfs_0] + _data_pdfs_10_2m1_36[_stride_pdfs_0 * x] + _data_pdfs_1m1_20_32[_stride_pdfs_0 * x] + _data_pdfs_1m1_20_39[_stride_pdfs_0 * x - _stride_pdfs_0] + _data_pdfs_1m1_2m1_316[_stride_pdfs_0 * x];
    double *RESTRICT _data_pdfs760dce667daab9ae = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_1 * neighbour_offset_y[dir] + _stride_pdfs_2 * z + _stride_pdfs_2 * neighbour_offset_z[dir] + _stride_pdfs_3 * f_in_inv_dir_idx[dir];
    uint8_t *RESTRICT _data_indexVector_116 = _data_indexVector + 16;
    uint8_t *RESTRICT _data_indexVector_124 = _data_indexVector + 24;
    uint8_t *RESTRICT _data_indexVector_132 = _data_indexVector + 32;
    double *RESTRICT _data_pdfs_10_200a5bfb2297cee9db = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + _stride_pdfs_3 * dir;
    _data_pdfs760dce667daab9ae[_stride_pdfs_0 * x + _stride_pdfs_0 * neighbour_offset_x[dir]] = -rho * (6.0 * ((double)(neighbour_offset_x[dir])) * *((double *)(&_data_indexVector_116[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x])) + 6.0 * ((double)(neighbour_offset_y[dir])) * *((double *)(&_data_indexVector_124[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x])) + 6.0 * ((double)(neighbour_offset_z[dir])) * *((double *)(&_data_indexVector_132[40 * blockDim.x * blockIdx.x + 40 * threadIdx.x]))) * weights[dir] + _data_pdfs_10_200a5bfb2297cee9db[_stride_pdfs_0 * x];
  }
}
} // namespace internal_dynamic_ubb_double_precisioncuda_boundary_Dynamic_UBB_double_precisionCUDA

// NOLINTEND(readability-non-const-parameter*)

#if defined(__clang__)
#if defined(__CUDA__)
#if defined(__CUDA_ARCH__)
// clang compiling CUDA code in device mode
#pragma clang diagnostic pop
#else
// clang compiling CUDA code in host mode
#pragma clang diagnostic pop
#endif
#endif
#elif defined(__GNUC__) or defined(__GNUG__)
#pragma GCC diagnostic pop
#elif defined(__HIPCC__)
#if defined(__NVCC_DIAG_PRAGMA_SUPPORT__)
#pragma nv_diagnostic pop
#else
#pragma pop
#endif
#endif

void Dynamic_UBB_double_precisionCUDA::run_impl(IBlock *block, IndexVectors::Type type, gpuStream_t stream) {
  auto *indexVectors = block->getData<IndexVectors>(indexVectorID);
  int32_t indexVectorSize = int32_c(indexVectors->indexVector(type).size());
  if (indexVectorSize == 0)
    return;

  auto pointer = indexVectors->pointerGpu(type);

  uint8_t *_data_indexVector = reinterpret_cast<uint8_t *>(pointer);

  auto pdfs = block->getData<gpu::GPUField<double>>(pdfsID);

  WALBERLA_ASSERT_GREATER_EQUAL(0, -int_c(pdfs->nrOfGhostLayers()))
  double *RESTRICT _data_pdfs = pdfs->dataAt(0, 0, 0, 0);
  const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
  const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
  const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
  const int64_t _stride_pdfs_3 = int64_t(1 * int64_t(pdfs->fStride()));
  dim3 _block(uint32_c(((256 < indexVectorSize) ? 256 : indexVectorSize)), uint32_c(1), uint32_c(1));
  dim3 _grid(uint32_c(((indexVectorSize) % (((256 < indexVectorSize) ? 256 : indexVectorSize)) == 0 ? (int64_t)(indexVectorSize) / (int64_t)(((256 < indexVectorSize) ? 256 : indexVectorSize)) : ((int64_t)(indexVectorSize) / (int64_t)(((256 < indexVectorSize) ? 256 : indexVectorSize))) + 1)), uint32_c(1), uint32_c(1));
  internal_dynamic_ubb_double_precisioncuda_boundary_Dynamic_UBB_double_precisionCUDA::dynamic_ubb_double_precisioncuda_boundary_Dynamic_UBB_double_precisionCUDA<<<_grid, _block, 0, stream>>>(_data_indexVector, _data_pdfs, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3, indexVectorSize);
}

void Dynamic_UBB_double_precisionCUDA::run(IBlock *block, gpuStream_t stream) {
  run_impl(block, IndexVectors::ALL, stream);
}

void Dynamic_UBB_double_precisionCUDA::inner(IBlock *block, gpuStream_t stream) {
  run_impl(block, IndexVectors::INNER, stream);
}

void Dynamic_UBB_double_precisionCUDA::outer(IBlock *block, gpuStream_t stream) {
  run_impl(block, IndexVectors::OUTER, stream);
}

} // namespace lbm
} // namespace walberla
