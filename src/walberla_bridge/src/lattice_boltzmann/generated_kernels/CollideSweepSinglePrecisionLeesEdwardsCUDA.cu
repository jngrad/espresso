#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \\file CollideSweepSinglePrecisionLeesEdwardsCUDA.cpp
//! \\author pystencils
//======================================================================================================================

// kernel generated with pystencils v1.2, lbmpy v1.2, lbmpy_walberla/pystencils_walberla from waLBerla commit 0c8b4b926c6979288fd8a6846d02ec0870e1fe41

#include <cmath>

#include "CollideSweepSinglePrecisionLeesEdwardsCUDA.h"
#include "core/DataTypes.h"
#include "core/Macros.h"

#define FUNC_PREFIX __global__

#if (defined WALBERLA_CXX_COMPILER_IS_GNU) || (defined WALBERLA_CXX_COMPILER_IS_CLANG)
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wfloat-equal"
#pragma GCC diagnostic ignored "-Wshadow"
#pragma GCC diagnostic ignored "-Wconversion"
#pragma GCC diagnostic ignored "-Wunused-variable"
#endif

#if (defined WALBERLA_CXX_COMPILER_IS_INTEL)
#pragma warning push
#pragma warning(disable : 1599)
#endif

using namespace std;

namespace walberla {
namespace pystencils {

namespace internal_collidesweepsingleprecisionleesedwardscuda_collidesweepsingleprecisionleesedwardscuda {
static FUNC_PREFIX __launch_bounds__(256) void collidesweepsingleprecisionleesedwardscuda_collidesweepsingleprecisionleesedwardscuda(float *RESTRICT const _data_force, float *RESTRICT _data_pdfs, int64_t const _size_force_0, int64_t const _size_force_1, int64_t const _size_force_2, int64_t const _stride_force_0, int64_t const _stride_force_1, int64_t const _stride_force_2, int64_t const _stride_force_3, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3, float grid_size, float omega_shear, float v_s) {
  if (blockDim.x * blockIdx.x + threadIdx.x < _size_force_0 && blockDim.y * blockIdx.y + threadIdx.y < _size_force_1 && blockDim.z * blockIdx.z + threadIdx.z < _size_force_2) {
    const int64_t ctr_0 = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t ctr_1 = blockDim.y * blockIdx.y + threadIdx.y;
    const int64_t ctr_2 = blockDim.z * blockIdx.z + threadIdx.z;
    float *RESTRICT _data_pdfs_10_20_36 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 6 * _stride_pdfs_3;
    const float xi_25 = _data_pdfs_10_20_36[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_314 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 14 * _stride_pdfs_3;
    const float xi_26 = _data_pdfs_10_20_314[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_38 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 8 * _stride_pdfs_3;
    const float xi_27 = _data_pdfs_10_20_38[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_39 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 9 * _stride_pdfs_3;
    const float xi_28 = _data_pdfs_10_20_39[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_318 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 18 * _stride_pdfs_3;
    const float xi_29 = _data_pdfs_10_20_318[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_32 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 2 * _stride_pdfs_3;
    const float xi_30 = _data_pdfs_10_20_32[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_33 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 3 * _stride_pdfs_3;
    const float xi_31 = _data_pdfs_10_20_33[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_315 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 15 * _stride_pdfs_3;
    const float xi_32 = _data_pdfs_10_20_315[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_310 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 10 * _stride_pdfs_3;
    const float xi_33 = _data_pdfs_10_20_310[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_force_10_20_31 = _data_force + _stride_force_1 * ctr_1 + _stride_force_2 * ctr_2 + _stride_force_3;
    const float xi_34 = _data_force_10_20_31[_stride_force_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_313 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 13 * _stride_pdfs_3;
    const float xi_35 = _data_pdfs_10_20_313[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_317 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 17 * _stride_pdfs_3;
    const float xi_36 = _data_pdfs_10_20_317[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_37 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 7 * _stride_pdfs_3;
    const float xi_37 = _data_pdfs_10_20_37[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_312 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 12 * _stride_pdfs_3;
    const float xi_38 = _data_pdfs_10_20_312[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_30 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2;
    const float xi_39 = _data_pdfs_10_20_30[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_311 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 11 * _stride_pdfs_3;
    const float xi_40 = _data_pdfs_10_20_311[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_force_10_20_32 = _data_force + _stride_force_1 * ctr_1 + _stride_force_2 * ctr_2 + 2 * _stride_force_3;
    const float xi_41 = _data_force_10_20_32[_stride_force_0 * ctr_0];
    float *RESTRICT _data_force_10_20_30 = _data_force + _stride_force_1 * ctr_1 + _stride_force_2 * ctr_2;
    const float xi_42 = _data_force_10_20_30[_stride_force_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_35 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 5 * _stride_pdfs_3;
    const float xi_43 = _data_pdfs_10_20_35[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_34 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 4 * _stride_pdfs_3;
    const float xi_44 = _data_pdfs_10_20_34[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_316 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + 16 * _stride_pdfs_3;
    const float xi_45 = _data_pdfs_10_20_316[_stride_pdfs_0 * ctr_0];
    float *RESTRICT _data_pdfs_10_20_31 = _data_pdfs + _stride_pdfs_1 * ctr_1 + _stride_pdfs_2 * ctr_2 + _stride_pdfs_3;
    const float xi_46 = _data_pdfs_10_20_31[_stride_pdfs_0 * ctr_0];
    const float xi_3 = xi_25;
    const float xi_4 = xi_41;
    const float xi_5 = xi_46;
    const float xi_6 = xi_30;
    const float xi_7 = xi_40;
    const float xi_8 = xi_29;
    const float xi_9 = xi_35;
    const float xi_10 = xi_39;
    const float xi_11 = xi_44;
    const float xi_12 = xi_31;
    const float xi_13 = xi_36;
    const float xi_14 = xi_42;
    const float xi_15 = xi_43;
    const float xi_16 = xi_26;
    const float xi_17 = xi_27;
    const float xi_18 = xi_38;
    const float xi_19 = xi_45;
    const float xi_20 = xi_32;
    const float xi_21 = xi_34;
    const float xi_22 = xi_33;
    const float xi_23 = xi_28;
    const float xi_24 = xi_37;
    const float xi_0 = ((1.0f) / (omega_shear * -0.25f + 2.0f));
    const float rr_0 = xi_0 * (omega_shear * -2.0f + 4.0f);
    const float vel0Term = xi_11 + xi_16 + xi_17 + xi_22 + xi_8;
    const float vel1Term = xi_20 + xi_24 + xi_5 + xi_7;
    const float vel2Term = xi_15 + xi_18 + xi_9;
    const float rho = vel0Term + vel1Term + vel2Term + xi_10 + xi_12 + xi_13 + xi_19 + xi_23 + xi_3 + xi_6;
    const float xi_1 = ((1.0f) / (rho));
    const float u_0 = xi_1 * xi_14 * 0.5f + xi_1 * (vel0Term + xi_12 * -1.0f + xi_13 * -1.0f + xi_23 * -1.0f + xi_24 * -1.0f + xi_9 * -1.0f);
    const float u_1 = xi_1 * xi_21 * 0.5f + xi_1 * (vel1Term + xi_17 + xi_18 * -1.0f + xi_19 * -1.0f + xi_22 * -1.0f + xi_23 * -1.0f + xi_6 * -1.0f);
    const float u_2 = xi_1 * xi_4 * 0.5f + xi_1 * (vel2Term + xi_13 * -1.0f + xi_16 + xi_19 * -1.0f + xi_20 * -1.0f + xi_3 * -1.0f + xi_7 + xi_8 * -1.0f);
    const float forceTerm_0 = omega_shear * u_0 * xi_14 * 0.5f + omega_shear * u_1 * xi_21 * 0.5f + omega_shear * u_2 * xi_4 * 0.5f + u_0 * xi_14 * -1.0f + u_1 * xi_21 * -1.0f + u_2 * xi_4 * -1.0f;
    const float forceTerm_1 = omega_shear * u_0 * xi_14 * 0.083333333333333329f + omega_shear * u_1 * xi_21 * -0.16666666666666666f + omega_shear * u_2 * xi_4 * 0.083333333333333329f + rr_0 * xi_21 * -0.083333333333333329f + u_0 * xi_14 * -0.16666666666666666f + u_1 * xi_21 * 0.33333333333333331f + u_2 * xi_4 * -0.16666666666666666f + xi_21 * 0.16666666666666666f;
    const float forceTerm_2 = omega_shear * u_0 * xi_14 * 0.083333333333333329f + omega_shear * u_1 * xi_21 * -0.16666666666666666f + omega_shear * u_2 * xi_4 * 0.083333333333333329f + rr_0 * xi_21 * 0.083333333333333329f + u_0 * xi_14 * -0.16666666666666666f + u_1 * xi_21 * 0.33333333333333331f + u_2 * xi_4 * -0.16666666666666666f + xi_21 * -0.16666666666666666f;
    const float forceTerm_3 = omega_shear * u_0 * xi_14 * -0.16666666666666666f + omega_shear * u_1 * xi_21 * 0.083333333333333329f + omega_shear * u_2 * xi_4 * 0.083333333333333329f + rr_0 * xi_14 * 0.083333333333333329f + u_0 * xi_14 * 0.33333333333333331f + u_1 * xi_21 * -0.16666666666666666f + u_2 * xi_4 * -0.16666666666666666f + xi_14 * -0.16666666666666666f;
    const float forceTerm_4 = omega_shear * u_0 * xi_14 * -0.16666666666666666f + omega_shear * u_1 * xi_21 * 0.083333333333333329f + omega_shear * u_2 * xi_4 * 0.083333333333333329f + rr_0 * xi_14 * -0.083333333333333329f + u_0 * xi_14 * 0.33333333333333331f + u_1 * xi_21 * -0.16666666666666666f + u_2 * xi_4 * -0.16666666666666666f + xi_14 * 0.16666666666666666f;
    const float forceTerm_5 = omega_shear * u_0 * xi_14 * 0.083333333333333329f + omega_shear * u_1 * xi_21 * 0.083333333333333329f + omega_shear * u_2 * xi_4 * -0.16666666666666666f + rr_0 * xi_4 * -0.083333333333333329f + u_0 * xi_14 * -0.16666666666666666f + u_1 * xi_21 * -0.16666666666666666f + u_2 * xi_4 * 0.33333333333333331f + xi_4 * 0.16666666666666666f;
    const float forceTerm_6 = omega_shear * u_0 * xi_14 * 0.083333333333333329f + omega_shear * u_1 * xi_21 * 0.083333333333333329f + omega_shear * u_2 * xi_4 * -0.16666666666666666f + rr_0 * xi_4 * 0.083333333333333329f + u_0 * xi_14 * -0.16666666666666666f + u_1 * xi_21 * -0.16666666666666666f + u_2 * xi_4 * 0.33333333333333331f + xi_4 * -0.16666666666666666f;
    const float forceTerm_7 = omega_shear * u_0 * xi_14 * -0.083333333333333329f + omega_shear * u_0 * xi_21 * 0.125f + omega_shear * u_1 * xi_14 * 0.125f + omega_shear * u_1 * xi_21 * -0.083333333333333329f + omega_shear * u_2 * xi_4 * 0.041666666666666664f + rr_0 * xi_14 * 0.041666666666666664f + rr_0 * xi_21 * -0.041666666666666664f + u_0 * xi_14 * 0.16666666666666666f + u_0 * xi_21 * -0.25f + u_1 * xi_14 * -0.25f + u_1 * xi_21 * 0.16666666666666666f + u_2 * xi_4 * -0.083333333333333329f + xi_14 * -0.083333333333333329f + xi_21 * 0.083333333333333329f;
    const float forceTerm_8 = omega_shear * u_0 * xi_14 * -0.083333333333333329f + omega_shear * u_0 * xi_21 * -0.125f + omega_shear * u_1 * xi_14 * -0.125f + omega_shear * u_1 * xi_21 * -0.083333333333333329f + omega_shear * u_2 * xi_4 * 0.041666666666666664f + rr_0 * xi_14 * -0.041666666666666664f + rr_0 * xi_21 * -0.041666666666666664f + u_0 * xi_14 * 0.16666666666666666f + u_0 * xi_21 * 0.25f + u_1 * xi_14 * 0.25f + u_1 * xi_21 * 0.16666666666666666f + u_2 * xi_4 * -0.083333333333333329f + xi_14 * 0.083333333333333329f + xi_21 * 0.083333333333333329f;
    const float forceTerm_9 = omega_shear * u_0 * xi_14 * -0.083333333333333329f + omega_shear * u_0 * xi_21 * -0.125f + omega_shear * u_1 * xi_14 * -0.125f + omega_shear * u_1 * xi_21 * -0.083333333333333329f + omega_shear * u_2 * xi_4 * 0.041666666666666664f + rr_0 * xi_14 * 0.041666666666666664f + rr_0 * xi_21 * 0.041666666666666664f + u_0 * xi_14 * 0.16666666666666666f + u_0 * xi_21 * 0.25f + u_1 * xi_14 * 0.25f + u_1 * xi_21 * 0.16666666666666666f + u_2 * xi_4 * -0.083333333333333329f + xi_14 * -0.083333333333333329f + xi_21 * -0.083333333333333329f;
    const float forceTerm_10 = omega_shear * u_0 * xi_14 * -0.083333333333333329f + omega_shear * u_0 * xi_21 * 0.125f + omega_shear * u_1 * xi_14 * 0.125f + omega_shear * u_1 * xi_21 * -0.083333333333333329f + omega_shear * u_2 * xi_4 * 0.041666666666666664f + rr_0 * xi_14 * -0.041666666666666664f + rr_0 * xi_21 * 0.041666666666666664f + u_0 * xi_14 * 0.16666666666666666f + u_0 * xi_21 * -0.25f + u_1 * xi_14 * -0.25f + u_1 * xi_21 * 0.16666666666666666f + u_2 * xi_4 * -0.083333333333333329f + xi_14 * 0.083333333333333329f + xi_21 * -0.083333333333333329f;
    const float forceTerm_11 = omega_shear * u_0 * xi_14 * 0.041666666666666664f + omega_shear * u_1 * xi_21 * -0.083333333333333329f + omega_shear * u_1 * xi_4 * -0.125f + omega_shear * u_2 * xi_21 * -0.125f + omega_shear * u_2 * xi_4 * -0.083333333333333329f + rr_0 * xi_21 * -0.041666666666666664f + rr_0 * xi_4 * -0.041666666666666664f + u_0 * xi_14 * -0.083333333333333329f + u_1 * xi_21 * 0.16666666666666666f + u_1 * xi_4 * 0.25f + u_2 * xi_21 * 0.25f + u_2 * xi_4 * 0.16666666666666666f + xi_21 * 0.083333333333333329f + xi_4 * 0.083333333333333329f;
    const float forceTerm_12 = omega_shear * u_0 * xi_14 * 0.041666666666666664f + omega_shear * u_1 * xi_21 * -0.083333333333333329f + omega_shear * u_1 * xi_4 * 0.125f + omega_shear * u_2 * xi_21 * 0.125f + omega_shear * u_2 * xi_4 * -0.083333333333333329f + rr_0 * xi_21 * 0.041666666666666664f + rr_0 * xi_4 * -0.041666666666666664f + u_0 * xi_14 * -0.083333333333333329f + u_1 * xi_21 * 0.16666666666666666f + u_1 * xi_4 * -0.25f + u_2 * xi_21 * -0.25f + u_2 * xi_4 * 0.16666666666666666f + xi_21 * -0.083333333333333329f + xi_4 * 0.083333333333333329f;
    const float forceTerm_13 = omega_shear * u_0 * xi_14 * -0.083333333333333329f + omega_shear * u_0 * xi_4 * 0.125f + omega_shear * u_1 * xi_21 * 0.041666666666666664f + omega_shear * u_2 * xi_14 * 0.125f + omega_shear * u_2 * xi_4 * -0.083333333333333329f + rr_0 * xi_14 * 0.041666666666666664f + rr_0 * xi_4 * -0.041666666666666664f + u_0 * xi_14 * 0.16666666666666666f + u_0 * xi_4 * -0.25f + u_1 * xi_21 * -0.083333333333333329f + u_2 * xi_14 * -0.25f + u_2 * xi_4 * 0.16666666666666666f + xi_14 * -0.083333333333333329f + xi_4 * 0.083333333333333329f;
    const float forceTerm_14 = omega_shear * u_0 * xi_14 * -0.083333333333333329f + omega_shear * u_0 * xi_4 * -0.125f + omega_shear * u_1 * xi_21 * 0.041666666666666664f + omega_shear * u_2 * xi_14 * -0.125f + omega_shear * u_2 * xi_4 * -0.083333333333333329f + rr_0 * xi_14 * -0.041666666666666664f + rr_0 * xi_4 * -0.041666666666666664f + u_0 * xi_14 * 0.16666666666666666f + u_0 * xi_4 * 0.25f + u_1 * xi_21 * -0.083333333333333329f + u_2 * xi_14 * 0.25f + u_2 * xi_4 * 0.16666666666666666f + xi_14 * 0.083333333333333329f + xi_4 * 0.083333333333333329f;
    const float forceTerm_15 = omega_shear * u_0 * xi_14 * 0.041666666666666664f + omega_shear * u_1 * xi_21 * -0.083333333333333329f + omega_shear * u_1 * xi_4 * 0.125f + omega_shear * u_2 * xi_21 * 0.125f + omega_shear * u_2 * xi_4 * -0.083333333333333329f + rr_0 * xi_21 * -0.041666666666666664f + rr_0 * xi_4 * 0.041666666666666664f + u_0 * xi_14 * -0.083333333333333329f + u_1 * xi_21 * 0.16666666666666666f + u_1 * xi_4 * -0.25f + u_2 * xi_21 * -0.25f + u_2 * xi_4 * 0.16666666666666666f + xi_21 * 0.083333333333333329f + xi_4 * -0.083333333333333329f;
    const float forceTerm_16 = omega_shear * u_0 * xi_14 * 0.041666666666666664f + omega_shear * u_1 * xi_21 * -0.083333333333333329f + omega_shear * u_1 * xi_4 * -0.125f + omega_shear * u_2 * xi_21 * -0.125f + omega_shear * u_2 * xi_4 * -0.083333333333333329f + rr_0 * xi_21 * 0.041666666666666664f + rr_0 * xi_4 * 0.041666666666666664f + u_0 * xi_14 * -0.083333333333333329f + u_1 * xi_21 * 0.16666666666666666f + u_1 * xi_4 * 0.25f + u_2 * xi_21 * 0.25f + u_2 * xi_4 * 0.16666666666666666f + xi_21 * -0.083333333333333329f + xi_4 * -0.083333333333333329f;
    const float forceTerm_17 = omega_shear * u_0 * xi_14 * -0.083333333333333329f + omega_shear * u_0 * xi_4 * -0.125f + omega_shear * u_1 * xi_21 * 0.041666666666666664f + omega_shear * u_2 * xi_14 * -0.125f + omega_shear * u_2 * xi_4 * -0.083333333333333329f + rr_0 * xi_14 * 0.041666666666666664f + rr_0 * xi_4 * 0.041666666666666664f + u_0 * xi_14 * 0.16666666666666666f + u_0 * xi_4 * 0.25f + u_1 * xi_21 * -0.083333333333333329f + u_2 * xi_14 * 0.25f + u_2 * xi_4 * 0.16666666666666666f + xi_14 * -0.083333333333333329f + xi_4 * -0.083333333333333329f;
    const float forceTerm_18 = omega_shear * u_0 * xi_14 * -0.083333333333333329f + omega_shear * u_0 * xi_4 * 0.125f + omega_shear * u_1 * xi_21 * 0.041666666666666664f + omega_shear * u_2 * xi_14 * 0.125f + omega_shear * u_2 * xi_4 * -0.083333333333333329f + rr_0 * xi_14 * -0.041666666666666664f + rr_0 * xi_4 * 0.041666666666666664f + u_0 * xi_14 * 0.16666666666666666f + u_0 * xi_4 * -0.25f + u_1 * xi_21 * -0.083333333333333329f + u_2 * xi_14 * -0.25f + u_2 * xi_4 * 0.16666666666666666f + xi_14 * 0.083333333333333329f + xi_4 * -0.083333333333333329f;
    const float u0Mu1 = u_0 + u_1 * -1.0f;
    const float u0Pu1 = u_0 + u_1;
    const float u1Pu2 = u_1 + u_2;
    const float u1Mu2 = u_1 + u_2 * -1.0f;
    const float u0Mu2 = u_0 + u_2 * -1.0f;
    const float u0Pu2 = u_0 + u_2;
    const float f_eq_common = rho * -1.0f * (u_0 * u_0) + rho * -1.0f * (u_1 * u_1) + rho * -1.0f * (u_2 * u_2) + rho;
    _data_pdfs_10_20_30[_stride_pdfs_0 * ctr_0] = forceTerm_0 + omega_shear * (f_eq_common * 0.33333333333333331f + xi_10 * -1.0f) + xi_10;
    _data_pdfs_10_20_31[_stride_pdfs_0 * ctr_0] = forceTerm_1 + omega_shear * (f_eq_common * 0.16666666666666666f + rho * (-0.1111111111111111f + 0.33333333333333331f * (u_1 * u_1)) + xi_5 * -0.5f + xi_6 * -0.5f) + rr_0 * (rho * u_1 * 0.16666666666666666f + xi_5 * -0.5f + xi_6 * 0.5f) + xi_5 + ((-1.0f <= grid_size * -1.0f + ((float)(ctr_1))) ? (rho * v_s * (u_0 * 2.0f + v_s) * 0.16666666666666666f) : (0.0f));
    _data_pdfs_10_20_32[_stride_pdfs_0 * ctr_0] = forceTerm_2 + omega_shear * (f_eq_common * 0.16666666666666666f + rho * (-0.1111111111111111f + 0.33333333333333331f * (u_1 * u_1)) + xi_5 * -0.5f + xi_6 * -0.5f) + rr_0 * (rho * u_1 * -0.16666666666666666f + xi_5 * 0.5f + xi_6 * -0.5f) + xi_6 + ((0.0f >= ((float)(ctr_1))) ? (rho * v_s * (u_0 * -2.0f + v_s) * 0.16666666666666666f) : (0.0f));
    _data_pdfs_10_20_33[_stride_pdfs_0 * ctr_0] = forceTerm_3 + omega_shear * (f_eq_common * 0.16666666666666666f + rho * (-0.1111111111111111f + 0.33333333333333331f * (u_0 * u_0)) + xi_11 * -0.5f + xi_12 * -0.5f) + rr_0 * (rho * u_0 * -0.16666666666666666f + xi_11 * 0.5f + xi_12 * -0.5f) + xi_12;
    _data_pdfs_10_20_34[_stride_pdfs_0 * ctr_0] = forceTerm_4 + omega_shear * (f_eq_common * 0.16666666666666666f + rho * (-0.1111111111111111f + 0.33333333333333331f * (u_0 * u_0)) + xi_11 * -0.5f + xi_12 * -0.5f) + rr_0 * (rho * u_0 * 0.16666666666666666f + xi_11 * -0.5f + xi_12 * 0.5f) + xi_11;
    _data_pdfs_10_20_35[_stride_pdfs_0 * ctr_0] = forceTerm_5 + omega_shear * (f_eq_common * 0.16666666666666666f + rho * (-0.1111111111111111f + 0.33333333333333331f * (u_2 * u_2)) + xi_15 * -0.5f + xi_3 * -0.5f) + rr_0 * (rho * u_2 * 0.16666666666666666f + xi_15 * -0.5f + xi_3 * 0.5f) + xi_15;
    _data_pdfs_10_20_36[_stride_pdfs_0 * ctr_0] = forceTerm_6 + omega_shear * (f_eq_common * 0.16666666666666666f + rho * (-0.1111111111111111f + 0.33333333333333331f * (u_2 * u_2)) + xi_15 * -0.5f + xi_3 * -0.5f) + rr_0 * (rho * u_2 * -0.16666666666666666f + xi_15 * 0.5f + xi_3 * -0.5f) + xi_3;
    _data_pdfs_10_20_37[_stride_pdfs_0 * ctr_0] = forceTerm_7 + omega_shear * (f_eq_common * 0.041666666666666664f + rho * (-0.013888888888888888f + 0.041666666666666664f * (u_2 * u_2) + 0.125f * (u0Mu1 * u0Mu1)) + xi_22 * -0.5f + xi_24 * -0.5f) + rr_0 * (rho * u0Mu1 * -0.083333333333333329f + xi_22 * 0.5f + xi_24 * -0.5f) + xi_24 + ((-1.0f <= grid_size * -1.0f + ((float)(ctr_1))) ? (rho * v_s * (u_0 * -2.0f + u_1 * 3.0f + v_s * -1.0f + 1.0f) * 0.083333333333333329f) : (0.0f));
    _data_pdfs_10_20_38[_stride_pdfs_0 * ctr_0] = forceTerm_8 + omega_shear * (f_eq_common * 0.041666666666666664f + rho * (-0.013888888888888888f + 0.041666666666666664f * (u_2 * u_2) + 0.125f * (u0Pu1 * u0Pu1)) + xi_17 * -0.5f + xi_23 * -0.5f) + rr_0 * (rho * u0Pu1 * 0.083333333333333329f + xi_17 * -0.5f + xi_23 * 0.5f) + xi_17 + ((-1.0f <= grid_size * -1.0f + ((float)(ctr_1))) ? (rho * v_s * (u_0 * 2.0f + u_1 * 3.0f + v_s + 1.0f) * -0.083333333333333329f) : (0.0f));
    _data_pdfs_10_20_39[_stride_pdfs_0 * ctr_0] = forceTerm_9 + omega_shear * (f_eq_common * 0.041666666666666664f + rho * (-0.013888888888888888f + 0.041666666666666664f * (u_2 * u_2) + 0.125f * (u0Pu1 * u0Pu1)) + xi_17 * -0.5f + xi_23 * -0.5f) + rr_0 * (rho * u0Pu1 * -0.083333333333333329f + xi_17 * 0.5f + xi_23 * -0.5f) + xi_23 + ((0.0f >= ((float)(ctr_1))) ? (rho * v_s * (u_0 * 2.0f + u_1 * 3.0f + v_s * -1.0f - 1.0f) * 0.083333333333333329f) : (0.0f));
    _data_pdfs_10_20_310[_stride_pdfs_0 * ctr_0] = forceTerm_10 + omega_shear * (f_eq_common * 0.041666666666666664f + rho * (-0.013888888888888888f + 0.041666666666666664f * (u_2 * u_2) + 0.125f * (u0Mu1 * u0Mu1)) + xi_22 * -0.5f + xi_24 * -0.5f) + rr_0 * (rho * u0Mu1 * 0.083333333333333329f + xi_22 * -0.5f + xi_24 * 0.5f) + xi_22 + ((0.0f >= ((float)(ctr_1))) ? (rho * v_s * (u_0 * 2.0f + u_1 * -3.0f + v_s * -1.0f + 1.0f) * 0.083333333333333329f) : (0.0f));
    _data_pdfs_10_20_311[_stride_pdfs_0 * ctr_0] = forceTerm_11 + omega_shear * (f_eq_common * 0.041666666666666664f + rho * (-0.013888888888888888f + 0.041666666666666664f * (u_0 * u_0) + 0.125f * (u1Pu2 * u1Pu2)) + xi_19 * -0.5f + xi_7 * -0.5f) + rr_0 * (rho * u1Pu2 * 0.083333333333333329f + xi_19 * 0.5f + xi_7 * -0.5f) + xi_7;
    _data_pdfs_10_20_312[_stride_pdfs_0 * ctr_0] = forceTerm_12 + omega_shear * (f_eq_common * 0.041666666666666664f + rho * (-0.013888888888888888f + 0.041666666666666664f * (u_0 * u_0) + 0.125f * (u1Mu2 * u1Mu2)) + xi_18 * -0.5f + xi_20 * -0.5f) + rr_0 * (rho * u1Mu2 * -0.083333333333333329f + xi_18 * -0.5f + xi_20 * 0.5f) + xi_18;
    _data_pdfs_10_20_313[_stride_pdfs_0 * ctr_0] = forceTerm_13 + omega_shear * (f_eq_common * 0.041666666666666664f + rho * (-0.013888888888888888f + 0.041666666666666664f * (u_1 * u_1) + 0.125f * (u0Mu2 * u0Mu2)) + xi_8 * -0.5f + xi_9 * -0.5f) + rr_0 * (rho * u0Mu2 * -0.083333333333333329f + xi_8 * 0.5f + xi_9 * -0.5f) + xi_9;
    _data_pdfs_10_20_314[_stride_pdfs_0 * ctr_0] = forceTerm_14 + omega_shear * (f_eq_common * 0.041666666666666664f + rho * (-0.013888888888888888f + 0.041666666666666664f * (u_1 * u_1) + 0.125f * (u0Pu2 * u0Pu2)) + xi_13 * -0.5f + xi_16 * -0.5f) + rr_0 * (rho * u0Pu2 * 0.083333333333333329f + xi_13 * 0.5f + xi_16 * -0.5f) + xi_16;
    _data_pdfs_10_20_315[_stride_pdfs_0 * ctr_0] = forceTerm_15 + omega_shear * (f_eq_common * 0.041666666666666664f + rho * (-0.013888888888888888f + 0.041666666666666664f * (u_0 * u_0) + 0.125f * (u1Mu2 * u1Mu2)) + xi_18 * -0.5f + xi_20 * -0.5f) + rr_0 * (rho * u1Mu2 * 0.083333333333333329f + xi_18 * 0.5f + xi_20 * -0.5f) + xi_20;
    _data_pdfs_10_20_316[_stride_pdfs_0 * ctr_0] = forceTerm_16 + omega_shear * (f_eq_common * 0.041666666666666664f + rho * (-0.013888888888888888f + 0.041666666666666664f * (u_0 * u_0) + 0.125f * (u1Pu2 * u1Pu2)) + xi_19 * -0.5f + xi_7 * -0.5f) + rr_0 * (rho * u1Pu2 * -0.083333333333333329f + xi_19 * -0.5f + xi_7 * 0.5f) + xi_19;
    _data_pdfs_10_20_317[_stride_pdfs_0 * ctr_0] = forceTerm_17 + omega_shear * (f_eq_common * 0.041666666666666664f + rho * (-0.013888888888888888f + 0.041666666666666664f * (u_1 * u_1) + 0.125f * (u0Pu2 * u0Pu2)) + xi_13 * -0.5f + xi_16 * -0.5f) + rr_0 * (rho * u0Pu2 * -0.083333333333333329f + xi_13 * -0.5f + xi_16 * 0.5f) + xi_13;
    _data_pdfs_10_20_318[_stride_pdfs_0 * ctr_0] = forceTerm_18 + omega_shear * (f_eq_common * 0.041666666666666664f + rho * (-0.013888888888888888f + 0.041666666666666664f * (u_1 * u_1) + 0.125f * (u0Mu2 * u0Mu2)) + xi_8 * -0.5f + xi_9 * -0.5f) + rr_0 * (rho * u0Mu2 * 0.083333333333333329f + xi_8 * -0.5f + xi_9 * 0.5f) + xi_8;
  }
}
} // namespace internal_collidesweepsingleprecisionleesedwardscuda_collidesweepsingleprecisionleesedwardscuda

void CollideSweepSinglePrecisionLeesEdwardsCUDA::run(IBlock *block, gpuStream_t stream) {
  auto force = block->getData<gpu::GPUField<float>>(forceID);
  auto pdfs = block->getData<gpu::GPUField<float>>(pdfsID);

  auto &grid_size = this->grid_size_;
  auto &v_s = this->v_s_;
  auto &omega_shear = this->omega_shear_;
  WALBERLA_ASSERT_GREATER_EQUAL(0, -int_c(force->nrOfGhostLayers()))
  float *RESTRICT const _data_force = force->dataAt(0, 0, 0, 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(0, -int_c(pdfs->nrOfGhostLayers()))
  float *RESTRICT _data_pdfs = pdfs->dataAt(0, 0, 0, 0);
  WALBERLA_ASSERT_EQUAL(pdfs->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->xSizeWithGhostLayer(), int64_t(int64_c(force->xSize()) + 0))
  const int64_t _size_force_0 = int64_t(int64_c(force->xSize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->ySizeWithGhostLayer(), int64_t(int64_c(force->ySize()) + 0))
  const int64_t _size_force_1 = int64_t(int64_c(force->ySize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->zSizeWithGhostLayer(), int64_t(int64_c(force->zSize()) + 0))
  const int64_t _size_force_2 = int64_t(int64_c(force->zSize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  const int64_t _stride_force_0 = int64_t(force->xStride());
  const int64_t _stride_force_1 = int64_t(force->yStride());
  const int64_t _stride_force_2 = int64_t(force->zStride());
  const int64_t _stride_force_3 = int64_t(1 * int64_t(force->fStride()));
  const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
  const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
  const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
  const int64_t _stride_pdfs_3 = int64_t(1 * int64_t(pdfs->fStride()));
  dim3 _block(uint32_t(((16 < _size_force_0) ? 16 : _size_force_0)), uint32_t(((1024 < ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))) ? 1024 : ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))), uint32_t(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))))));
  dim3 _grid(uint32_t(((_size_force_0) % (((16 < _size_force_0) ? 16 : _size_force_0)) == 0 ? (int64_t)(_size_force_0) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)) : ((int64_t)(_size_force_0) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))) + 1)), uint32_t(((_size_force_1) % (((1024 < ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))) ? 1024 : ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))) == 0 ? (int64_t)(_size_force_1) / (int64_t)(((1024 < ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))) ? 1024 : ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))) : ((int64_t)(_size_force_1) / (int64_t)(((1024 < ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))) ? 1024 : ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) + 1)), uint32_t(((_size_force_2) % (((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))))) == 0 ? (int64_t)(_size_force_2) / (int64_t)(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))))) : ((int64_t)(_size_force_2) / (int64_t)(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))))))) + 1)));
  internal_collidesweepsingleprecisionleesedwardscuda_collidesweepsingleprecisionleesedwardscuda::collidesweepsingleprecisionleesedwardscuda_collidesweepsingleprecisionleesedwardscuda<<<_grid, _block, 0, stream>>>(_data_force, _data_pdfs, _size_force_0, _size_force_1, _size_force_2, _stride_force_0, _stride_force_1, _stride_force_2, _stride_force_3, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3, grid_size, omega_shear, v_s);
}

void CollideSweepSinglePrecisionLeesEdwardsCUDA::runOnCellInterval(const shared_ptr<StructuredBlockStorage> &blocks, const CellInterval &globalCellInterval, cell_idx_t ghostLayers, IBlock *block, gpuStream_t stream) {
  CellInterval ci = globalCellInterval;
  CellInterval blockBB = blocks->getBlockCellBB(*block);
  blockBB.expand(ghostLayers);
  ci.intersect(blockBB);
  blocks->transformGlobalToBlockLocalCellInterval(ci, *block);
  if (ci.empty())
    return;

  auto force = block->getData<gpu::GPUField<float>>(forceID);
  auto pdfs = block->getData<gpu::GPUField<float>>(pdfsID);

  auto &grid_size = this->grid_size_;
  auto &v_s = this->v_s_;
  auto &omega_shear = this->omega_shear_;
  WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(force->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(force->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(force->nrOfGhostLayers()))
  float *RESTRICT const _data_force = force->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()))
  WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()))
  float *RESTRICT _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
  WALBERLA_ASSERT_EQUAL(pdfs->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->xSizeWithGhostLayer(), int64_t(int64_c(ci.xSize()) + 0))
  const int64_t _size_force_0 = int64_t(int64_c(ci.xSize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->ySizeWithGhostLayer(), int64_t(int64_c(ci.ySize()) + 0))
  const int64_t _size_force_1 = int64_t(int64_c(ci.ySize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  WALBERLA_ASSERT_GREATER_EQUAL(force->zSizeWithGhostLayer(), int64_t(int64_c(ci.zSize()) + 0))
  const int64_t _size_force_2 = int64_t(int64_c(ci.zSize()) + 0);
  WALBERLA_ASSERT_EQUAL(force->layout(), field::fzyx)
  const int64_t _stride_force_0 = int64_t(force->xStride());
  const int64_t _stride_force_1 = int64_t(force->yStride());
  const int64_t _stride_force_2 = int64_t(force->zStride());
  const int64_t _stride_force_3 = int64_t(1 * int64_t(force->fStride()));
  const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
  const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
  const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
  const int64_t _stride_pdfs_3 = int64_t(1 * int64_t(pdfs->fStride()));
  dim3 _block(uint32_t(((16 < _size_force_0) ? 16 : _size_force_0)), uint32_t(((1024 < ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))) ? 1024 : ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))), uint32_t(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))))));
  dim3 _grid(uint32_t(((_size_force_0) % (((16 < _size_force_0) ? 16 : _size_force_0)) == 0 ? (int64_t)(_size_force_0) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)) : ((int64_t)(_size_force_0) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))) + 1)), uint32_t(((_size_force_1) % (((1024 < ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))) ? 1024 : ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))) == 0 ? (int64_t)(_size_force_1) / (int64_t)(((1024 < ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))) ? 1024 : ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))) : ((int64_t)(_size_force_1) / (int64_t)(((1024 < ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))) ? 1024 : ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) + 1)), uint32_t(((_size_force_2) % (((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))))) == 0 ? (int64_t)(_size_force_2) / (int64_t)(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))))) : ((int64_t)(_size_force_2) / (int64_t)(((64 < ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))))) ? 64 : ((_size_force_2 < ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0))))))) ? _size_force_2 : ((int64_t)(256) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0) * ((_size_force_1 < 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))) ? _size_force_1 : 16 * ((int64_t)(16) / (int64_t)(((16 < _size_force_0) ? 16 : _size_force_0)))))))))) + 1)));
  internal_collidesweepsingleprecisionleesedwardscuda_collidesweepsingleprecisionleesedwardscuda::collidesweepsingleprecisionleesedwardscuda_collidesweepsingleprecisionleesedwardscuda<<<_grid, _block, 0, stream>>>(_data_force, _data_pdfs, _size_force_0, _size_force_1, _size_force_2, _stride_force_0, _stride_force_1, _stride_force_2, _stride_force_3, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3, grid_size, omega_shear, v_s);
}

} // namespace pystencils
} // namespace walberla

#if (defined WALBERLA_CXX_COMPILER_IS_GNU) || (defined WALBERLA_CXX_COMPILER_IS_CLANG)
#pragma GCC diagnostic pop
#endif

#if (defined WALBERLA_CXX_COMPILER_IS_INTEL)
#pragma warning pop
#endif