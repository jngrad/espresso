/*
 * Copyright (C) 2024 The ESPResSo project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "cuda/init.hpp"
#include "cuda/utils.cuh"
#include "cuda/utils.hpp"
#include "errorhandling.hpp"

#include <cassert>
#include <cstdlib>
#include <limits>
#include <string>

#include <boost/test/unit_test.hpp>

boost::test_tools::assertion_result has_gpu(boost::unit_test::test_unit_id) {
  int n_devices = 0;
  hipGetDeviceCount(&n_devices);
  if (n_devices > 0) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    if (prop.major >= 3) {
      return true;
    }
  }
  return false;
}

static int fatal_error_counter = 0;

static void increment_counter() noexcept { ++fatal_error_counter; }

void gpu_interface_test() {
  fatal_error_counter = 0;
  auto local_error_counter = 0;
  {
    std::string const what = "message 1";
    try {
      throw cuda_fatal_error(what);
    } catch (cuda_fatal_error &err) {
      BOOST_CHECK_EQUAL(err.what(), what);
      BOOST_CHECK_EQUAL(err.get_terminate(), &errexit);
      err.set_terminate(nullptr);
      BOOST_CHECK_EQUAL(err.get_terminate(), nullptr);
      err.set_terminate(increment_counter);
      BOOST_CHECK_EQUAL(err.get_terminate(), &increment_counter);
      BOOST_CHECK_EQUAL(fatal_error_counter, local_error_counter);
    }
    ++local_error_counter;
    BOOST_REQUIRE_EQUAL(fatal_error_counter, local_error_counter);
  }
  {
    auto error_caught = false;
    auto const block = dim3{1, 2, 3};
    auto const grid = dim3{4, 5, 6};
    // should not throw
    cuda_check_errors_exit(block, grid, "", "", 0u);
    try {
      // trigger non-sticky CUDA error
      hipSetDevice(-1);
      // should clear the CUDA error flag and throw a fatal error
      cuda_check_errors_exit(block, grid, "hipSetDevice()", "filename.cu", 4u);
    } catch (cuda_fatal_error &err) {
      error_caught = true;
      err.set_terminate(increment_counter);
      std::string const what =
          "CUDA error: \"invalid device ordinal\" calling hipSetDevice() with "
          "block: [1,2,3], grid: [4,5,6] in filename.cu:4";
      BOOST_CHECK_EQUAL(fatal_error_counter, local_error_counter);
      BOOST_CHECK_EQUAL(err.what(), what);
      BOOST_CHECK_EQUAL(hipGetLastError(), hipSuccess);
    }
    ++local_error_counter;
    BOOST_REQUIRE(error_caught);
    BOOST_REQUIRE_EQUAL(fatal_error_counter, local_error_counter);
  }
  {
    auto error_caught = false;
    // should not throw
    cuda_safe_mem_exit(hipSuccess, "", 0u);
    try {
      // trigger non-sticky CUDA error
      hipSetDevice(-1);
      // should throw
      cuda_safe_mem_exit(hipSuccess, "function_name()", 4u);
    } catch (cuda_fatal_error &err) {
      error_caught = true;
      err.set_terminate(increment_counter);
      std::string const what =
          "CUDA error: \"invalid device ordinal\" in function_name():4. Error "
          "found during memory operation. Possibly however from a failed "
          "operation before the memory operation";
      BOOST_CHECK_EQUAL(fatal_error_counter, local_error_counter);
      BOOST_CHECK_EQUAL(err.what(), what);
    }
    ++local_error_counter;
    BOOST_REQUIRE(error_caught);
    BOOST_REQUIRE_EQUAL(fatal_error_counter, local_error_counter);
  }
  {
    auto error_caught = false;
    try {
      cuda_safe_mem_exit(cudaErrorNotPermitted, "function_name()", 4u);
    } catch (cuda_fatal_error &err) {
      error_caught = true;
      err.set_terminate(increment_counter);
      std::string const what = "CUDA error: \"operation not permitted\" during "
                               "memory operation in function_name():4";
      BOOST_CHECK_EQUAL(fatal_error_counter, local_error_counter);
      BOOST_CHECK_EQUAL(err.what(), what);
    }
    ++local_error_counter;
    BOOST_REQUIRE(error_caught);
    BOOST_REQUIRE_EQUAL(fatal_error_counter, local_error_counter);
  }
  {
    auto error_caught = false;
    try {
      cuda_safe_mem_exit(hipErrorInvalidValue, "function_name()", 4u);
    } catch (cuda_fatal_error &err) {
      error_caught = true;
      err.set_terminate(increment_counter);
      std::string const what =
          "CUDA error: \"invalid argument\" during memory operation in "
          "function_name():4. You may have tried to allocate zero memory";
      BOOST_CHECK_EQUAL(fatal_error_counter, local_error_counter);
      BOOST_CHECK_EQUAL(err.what(), what);
    }
    ++local_error_counter;
    BOOST_REQUIRE(error_caught);
    BOOST_REQUIRE_EQUAL(fatal_error_counter, local_error_counter);
  }
  {
    BOOST_REQUIRE_EQUAL(stream[0], nullptr);
    auto error_caught = false;
    cuda_init(); // allocate
    BOOST_REQUIRE_NE(stream[0], nullptr);
    cuda_set_device(0); // reallocate, may or may not result in the same pointer
    BOOST_REQUIRE_NE(stream[0], nullptr);
    auto const old_stream = stream[0];
    try {
      cuda_set_device(-1); // fail to reallocate, pointer remains the same
    } catch (cuda_runtime_error_cuda const &err) {
      error_caught = true;
      std::string const what = "CUDA error: invalid device ordinal";
      BOOST_CHECK_EQUAL(err.what(), what);
    }
    BOOST_REQUIRE(error_caught);
    BOOST_REQUIRE_EQUAL(stream[0], old_stream);
  }
  {
    BOOST_REQUIRE_GE(cuda_get_n_gpus(), 1);
    char gpu_name_buffer[260] = {'\0'};
    cuda_get_gpu_name(0, gpu_name_buffer);
    for (int i = 255; i < 260; ++i) {
      BOOST_REQUIRE_EQUAL(gpu_name_buffer[i], '\0');
    }
  }
}
