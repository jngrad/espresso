/*
 * Copyright (C) 2024 The ESPResSo project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#define BOOST_TEST_MODULE cuda test
#define BOOST_TEST_DYN_LINK
#include <boost/test/unit_test.hpp>

#include "cuda/init.hpp"
#include "cuda/utils.cuh"
#include "cuda/utils.hpp"
#include "errorhandling.hpp"

#include "cuda/CudaHostAllocator.hpp"

#include <hip/hip_runtime.h>

#include <boost/test/unit_test.hpp>

#include <cstddef>
#include <optional>
#include <string>

boost::test_tools::assertion_result has_gpu(boost::unit_test::test_unit_id) {
  int n_devices = 0;
  hipGetDeviceCount(&n_devices);
  if (n_devices > 0) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    if (prop.major >= 3) {
      return true;
    }
  }
  return false;
}

std::optional<std::string> read_pending_cuda_errors() {
  auto const CU_err = hipGetLastError();
  if (CU_err != hipSuccess) {
    auto const message = std::string(hipGetErrorString(CU_err));
    return {"There is a pending CUDA error: \"" + message + "\""};
  }
  return std::nullopt;
}

void setup() {}
void teardown() {
  auto error = read_pending_cuda_errors();
  BOOST_REQUIRE_MESSAGE(not error.has_value(), error.value_or(""));
}

namespace Testing::non_sticky_cuda_error {

/** @brief Trigger a non-sticky CUDA error for testing purposes. */
void trigger() { hipSetDevice(-1); }

/** @brief Clear a non-sticky CUDA error raised by @ref trigger. */
void clear() {
  auto const error_code = hipGetLastError();
  BOOST_REQUIRE_MESSAGE(error_code == hipSuccess or
                            error_code == hipErrorInvalidDevice,
                        "An unexpected CUDA error was pending!");
}

} // namespace Testing::non_sticky_cuda_error

static auto fixture = boost::unit_test::fixture(&setup, &teardown);

BOOST_AUTO_TEST_SUITE(suite, *boost::unit_test::precondition(has_gpu))

BOOST_AUTO_TEST_CASE(gpu_fixture, *fixture) {
  {
    auto error = read_pending_cuda_errors();
    BOOST_REQUIRE(not error.has_value());
  }
  {
    // check we can raise and clear non-sticky CUDA errors
    Testing::non_sticky_cuda_error::trigger();
    Testing::non_sticky_cuda_error::clear();
    auto error = read_pending_cuda_errors();
    BOOST_REQUIRE(not error.has_value());
  }
  {
    // check fixture can handle the default non-sticky CUDA error
    Testing::non_sticky_cuda_error::trigger();
    auto ref_what{"There is a pending CUDA error: \"invalid device ordinal\""};
    auto error = read_pending_cuda_errors();
    BOOST_REQUIRE(error.has_value());
    BOOST_REQUIRE_EQUAL(error.value(), ref_what);
    // sticky error should have been cleared
    error = read_pending_cuda_errors();
    BOOST_REQUIRE(not error.has_value());
  }
  {
    // check fixture can handle a custom non-sticky CUDA error
    hipHostMalloc(nullptr, std::size_t(0u));
    auto ref_what{"There is a pending CUDA error: \"invalid argument\""};
    auto error = read_pending_cuda_errors();
    BOOST_REQUIRE(error.has_value());
    BOOST_REQUIRE_EQUAL(error.value(), ref_what);
    // sticky error should have been cleared
    error = read_pending_cuda_errors();
    BOOST_REQUIRE(not error.has_value());
  }
}

static int fatal_error_counter = 0;
static void increment_counter() noexcept { ++fatal_error_counter; }

BOOST_AUTO_TEST_CASE(gpu_interface, *fixture) {
  fatal_error_counter = 0;
  auto local_error_counter = 0;
  {
    std::string const what = "message 1";
    try {
      throw cuda_fatal_error(what);
    } catch (cuda_fatal_error &err) {
      BOOST_CHECK_EQUAL(err.what(), what);
      BOOST_CHECK_EQUAL(err.get_terminate(), &errexit);
      err.set_terminate(nullptr);
      BOOST_CHECK_EQUAL(err.get_terminate(), nullptr);
      err.set_terminate(increment_counter);
      BOOST_CHECK_EQUAL(err.get_terminate(), &increment_counter);
      BOOST_CHECK_EQUAL(fatal_error_counter, local_error_counter);
    }
    ++local_error_counter;
    BOOST_REQUIRE_EQUAL(fatal_error_counter, local_error_counter);
  }
  {
    auto error_caught = false;
    auto const block = dim3{1, 2, 3};
    auto const grid = dim3{4, 5, 6};
    // should not throw
    cuda_check_errors_exit(block, grid, "", "", 0u);
    try {
      Testing::non_sticky_cuda_error::trigger();
      // should clear the CUDA error flag and throw a fatal error
      cuda_check_errors_exit(block, grid, "hipSetDevice()", "filename.cu", 4u);
    } catch (cuda_fatal_error &err) {
      error_caught = true;
      err.set_terminate(increment_counter);
      std::string const what =
          "CUDA error: \"invalid device ordinal\" while calling "
          "hipSetDevice() with block: <1,2,3>, grid: <4,5,6> in filename.cu:4";
      BOOST_CHECK_EQUAL(fatal_error_counter, local_error_counter);
      BOOST_CHECK_EQUAL(err.what(), what);
      BOOST_CHECK_EQUAL(hipGetLastError(), hipSuccess);
    }
    ++local_error_counter;
    BOOST_REQUIRE(error_caught);
    BOOST_REQUIRE_EQUAL(fatal_error_counter, local_error_counter);
  }
  {
    auto error_caught = false;
    // should not throw
    cuda_safe_mem_exit(hipSuccess, "", 0u);
    try {
      Testing::non_sticky_cuda_error::trigger();
      // should throw
      cuda_safe_mem_exit(hipSuccess, "filename.cu", 4u);
    } catch (cuda_fatal_error &err) {
      error_caught = true;
      err.set_terminate(increment_counter);
      std::string const what =
          "CUDA error: \"invalid device ordinal\" in filename.cu:4. Error "
          "found during memory operation. Possibly however from a failed "
          "operation before the memory operation";
      BOOST_CHECK_EQUAL(fatal_error_counter, local_error_counter);
      BOOST_CHECK_EQUAL(err.what(), what);
    }
    ++local_error_counter;
    BOOST_REQUIRE(error_caught);
    BOOST_REQUIRE_EQUAL(fatal_error_counter, local_error_counter);
  }
  {
    auto error_caught = false;
    try {
      cuda_safe_mem_exit(cudaErrorNotPermitted, "filename.cu", 4u);
    } catch (cuda_fatal_error &err) {
      error_caught = true;
      err.set_terminate(increment_counter);
      std::string const what = "CUDA error: \"operation not permitted\" during "
                               "memory operation in filename.cu:4";
      BOOST_CHECK_EQUAL(fatal_error_counter, local_error_counter);
      BOOST_CHECK_EQUAL(err.what(), what);
    }
    ++local_error_counter;
    BOOST_REQUIRE(error_caught);
    BOOST_REQUIRE_EQUAL(fatal_error_counter, local_error_counter);
  }
  {
    auto error_caught = false;
    try {
      cuda_safe_mem_exit(hipErrorInvalidValue, "function_name()", 4u);
    } catch (cuda_fatal_error &err) {
      error_caught = true;
      err.set_terminate(increment_counter);
      std::string const what =
          "CUDA error: \"invalid argument\" during memory operation in "
          "function_name():4. You may have tried to allocate zero memory";
      BOOST_CHECK_EQUAL(fatal_error_counter, local_error_counter);
      BOOST_CHECK_EQUAL(err.what(), what);
    }
    ++local_error_counter;
    BOOST_REQUIRE(error_caught);
    BOOST_REQUIRE_EQUAL(fatal_error_counter, local_error_counter);
  }
  {
    BOOST_REQUIRE_EQUAL(stream[0], nullptr);
    auto error_caught = false;
    cuda_init(); // allocate
    BOOST_REQUIRE_NE(stream[0], nullptr);
    cuda_set_device(0); // reallocate, may or may not result in the same pointer
    BOOST_REQUIRE_NE(stream[0], nullptr);
    auto const old_stream = stream[0];
    try {
      cuda_set_device(-1); // fail to reallocate, pointer remains the same
    } catch (cuda_runtime_error_cuda const &err) {
      error_caught = true;
      std::string const what = "CUDA error: invalid device ordinal";
      BOOST_CHECK_EQUAL(err.what(), what);
    }
    BOOST_REQUIRE(error_caught);
    BOOST_REQUIRE_EQUAL(stream[0], old_stream);
  }
  {
    BOOST_REQUIRE_GE(cuda_get_n_gpus(), 1);
    char gpu_name_buffer[260] = {'\0'};
    cuda_get_gpu_name(0, gpu_name_buffer);
    for (int i = 255; i < 260; ++i) {
      BOOST_REQUIRE_EQUAL(gpu_name_buffer[i], '\0');
    }
  }
}

BOOST_AUTO_TEST_SUITE_END()
