/*
 * Copyright (C) 2010-2022 The ESPResSo project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "errorhandling.hpp"

#include "utils.cuh"
#include "utils.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>
#include <sstream>
#include <string>
#include <utility>

hipStream_t stream[1];

void cuda_check_errors_exit(const dim3 &block, const dim3 &grid,
                            const char *function, const char *file,
                            unsigned int line) {
  hipError_t CU_err = hipGetLastError();
  if (CU_err != hipSuccess) {
    std::stringstream message;
    message << "CUDA error: \"" << hipGetErrorString(CU_err) << "\" "
            << "calling " << function << " with "
            << "block: [" << block.x << "," << block.y << "," << block.z
            << "], "
            << "grid: [" << grid.x << "," << grid.y << "," << grid.z << "] "
            << "in " << file << ":" << line;
    throw cuda_fatal_error(message.str());
  }
}

void cuda_safe_mem_exit(hipError_t CU_err, const char *file,
                        unsigned int line) {
  if (CU_err != hipSuccess) {
    std::stringstream message;
    message << "CUDA error: \"" << hipGetErrorString(CU_err)
            << "\" during memory operation in " << file << ":" << line;
    if (CU_err == hipErrorInvalidValue)
      message << ". You may have tried to allocate zero memory";
    throw cuda_fatal_error(message.str());
  }
  {
    CU_err = hipGetLastError();
    if (CU_err != hipSuccess) {
      std::stringstream message;
      message << "CUDA error: \"" << hipGetErrorString(CU_err) << "\" in "
              << file << ":" << line << ". Error found during memory operation"
              << ". Possibly however from a failed operation before the memory "
                 "operation";
      throw cuda_fatal_error(message.str());
    }
  }
}

cuda_fatal_error::cuda_fatal_error(std::string msg)
    : m_msg(std::move(msg)), m_terminate_handler(&errexit) {}

void cuda_fatal_error::terminate() noexcept {
  ((m_terminate_handler == nullptr) ? &std::abort : m_terminate_handler)();
}
