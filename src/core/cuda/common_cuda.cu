/*
 * Copyright (C) 2010-2022 The ESPResSo project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "errorhandling.hpp"

#include "utils.cuh"
#include "utils.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>
#include <sstream>
#include <string>
#include <utility>

hipStream_t stream[1];

static std::basic_ostream<char> &operator<<(std::basic_ostream<char> &os,
                                            const dim3 &dim) {
  os << "<" << dim.x << "," << dim.y << "," << dim.z << ">";
  return os;
}

static std::basic_ostream<char> &operator<<(std::basic_ostream<char> &os,
                                            hipError_t CU_err) {
  os << "CUDA error: \"" << hipGetErrorString(CU_err) << "\"";
  return os;
}

void cuda_check_errors_exit(const dim3 &block, const dim3 &grid,
                            const char *function, const char *file,
                            unsigned int line) {
  hipError_t CU_err = hipGetLastError();
  if (CU_err != hipSuccess) {
    std::stringstream message;
    message << CU_err << " while calling " << function
            << " with block: " << block << ", grid: " << grid << " in " << file
            << ":" << line;
    throw cuda_fatal_error(message.str());
  }
}

void cuda_safe_mem_exit(hipError_t CU_err, const char *file,
                        unsigned int line) {
  if (CU_err != hipSuccess) {
    std::stringstream message;
    message << CU_err << " during memory operation in " << file << ":" << line;
    if (CU_err == hipErrorInvalidValue)
      message << ". You may have tried to allocate zero memory";
    throw cuda_fatal_error(message.str());
  }
  {
    CU_err = hipGetLastError();
    if (CU_err != hipSuccess) {
      std::stringstream message;
      message << CU_err << " in " << file << ":" << line << ". Error found "
              << "during memory operation. Possibly however from a failed "
                 "operation before the memory operation";
      throw cuda_fatal_error(message.str());
    }
  }
}

cuda_fatal_error::cuda_fatal_error(std::string msg)
    : m_msg(std::move(msg)), m_terminate_handler(&errexit) {}

void cuda_fatal_error::terminate() noexcept {
  if (m_terminate_handler == nullptr or m_terminate_handler == errexit) {
    fprintf(stderr, "%s\n", what());
  }
  ((m_terminate_handler == nullptr) ? &std::abort : m_terminate_handler)();
}
